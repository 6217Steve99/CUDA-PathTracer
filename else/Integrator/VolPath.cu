#include "hip/hip_runtime.h"
#include "VolPath.h"
#include "../CudaTools.h"

//**************************VolPath Integrator*********************
__global__ void Volpath(int iter, int maxDepth) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	//init seed
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	//start
	float offsetx = uniform(rng) - 0.5f;
	float offsety = uniform(rng) - 0.5f;
	float unuse;
	float2 aperture = UniformSampleDisk(uniform(rng), uniform(rng), unuse);//for dof
	Ray ray = kernel_Resource.kernel_camera->GeneratePrimaryRay(x + offsetx, y + offsety, aperture);
	ray.tmin = kernel_Resource.kernel_epsilon;
	ray.medium = kernel_Resource.kernel_camera->medium == -1 ? nullptr : &kernel_Resource.kernel_mediums[kernel_Resource.kernel_camera->medium];

	float3 Li = make_float3(0.f, 0.f, 0.f);
	float3 beta = make_float3(1.f, 1.f, 1.f);
	Ray r = ray;
	Intersection isect;
	bool specular = false;
	for (int bounces = 0; bounces < maxDepth; ++bounces) {
		if (!Intersect(r, &isect)) {
			if ((bounces == 0 || specular) && kernel_Resource.kernel_infinite->isvalid)
				Li += beta * kernel_Resource.kernel_infinite->Le(r.destination);
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.nor;
		float2 uv = isect.uv;
		float3 dpdu = isect.dpdu;

		float sampledDist;
		bool sampledMedium = false;
		if (r.medium) {
			if (r.medium->type == MT_HOMOGENEOUS)
				beta *= r.medium->homogeneous.Sample(r, uniform, rng, sampledDist, sampledMedium);
			else
				beta *= r.medium->heterogeneous.Sample(r, uniform, rng, sampledDist, sampledMedium);
		}
		if (IsBlack(beta)) break;
		if (sampledMedium) {
			//TODO:������Ҫ�Բ���
			bool inf = false;
			float u = uniform(rng);
			float choicePdf;
			int idx = LookUpLightDistribution(u, choicePdf);
			if (idx == kernel_Resource.kernel_light_size) inf = true;
			float3 samplePos = r(sampledDist);
			float2 u1 = make_float2(uniform(rng), uniform(rng));
			float3 radiance, lightNor;
			Ray shadowRay;
			float lightPdf;
			if (!inf)
				kernel_Resource.kernel_lights[idx].SampleLight(samplePos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_Resource.kernel_epsilon);
			else
				kernel_Resource.kernel_infinite->SampleLight(samplePos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_Resource.kernel_epsilon);
			shadowRay.medium = r.medium;
			float3 tr = Tr(shadowRay, uniform, rng);
			float phase, unuse;
			r.medium->Phase(-r.destination, shadowRay.destination, phase, unuse);

			if (!IsBlack(radiance))
				Li += tr * beta * phase * radiance / (lightPdf * choicePdf);

			float pdf;
			float2 phaseU = make_float2(uniform(rng), uniform(rng));
			float3 dir;
			r.medium->SamplePhase(phaseU, dir, phase, pdf);
			r = Ray(samplePos, dir, r.medium, kernel_Resource.kernel_epsilon);
			specular = false;
		}
		else {
			if (bounces == 0 || specular) {
				if (isect.lightIdx != -1) {
					float3 tr = { 1.f, 1.f, 1.f };
					if (r.medium) {
						if (r.medium->type == MT_HOMOGENEOUS)
							tr = r.medium->homogeneous.Tr(r, uniform, rng);
						else
							tr = r.medium->heterogeneous.Tr(r, uniform, rng);
					}
					Li += tr * beta * kernel_Resource.kernel_lights[isect.lightIdx].Le(nor, -r.destination);
					break;
				}
			}

			if (isect.matIdx == -1) {
				bounces--;
				Medium* m = dot(r.destination, isect.nor) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_Resource.kernel_mediums[isect.mediumOutside])
					: (isect.mediumInside == -1 ? nullptr : &kernel_Resource.kernel_mediums[isect.mediumInside]);
				r = Ray(pos, r.destination, m, kernel_Resource.kernel_epsilon);

				continue;
			}

			Material material = kernel_Resource.kernel_materials[isect.matIdx];
			//direct light with multiple importance sampling
			if (!IsDelta(material.type)) {
				float3 Ld = make_float3(0.f, 0.f, 0.f);
				bool inf = false;
				float u = uniform(rng);
				float choicePdf;
				int idx = LookUpLightDistribution(u, choicePdf);
				if (idx == kernel_Resource.kernel_light_size) inf = true;
				float2 u1 = make_float2(uniform(rng), uniform(rng));
				float3 radiance, lightNor;
				Ray shadowRay;
				float lightPdf;
				if (!inf)
					kernel_Resource.kernel_lights[idx].SampleLight(pos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_Resource.kernel_epsilon);
				else
					kernel_Resource.kernel_infinite->SampleLight(pos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_Resource.kernel_epsilon);
				shadowRay.medium = r.medium;

				if (!IsBlack(radiance)) {
					float3 fr;
					float samplePdf;

					//Fr(material, -r.destination, shadowRay.destination, nor, uv, dpdu, uniform(rng), fr, samplePdf);
					Fr(material, -r.destination, shadowRay.destination, nor, uv, dpdu, fr, samplePdf);
					float3 tr = Tr(shadowRay, uniform, rng);

					float weight = PowerHeuristic(1, lightPdf * choicePdf, 1, samplePdf);
					Ld += weight * tr * fr * radiance * fabs(dot(nor, shadowRay.destination)) / (lightPdf * choicePdf);
				}

				float3 us = make_float3(uniform(rng), uniform(rng), uniform(rng));
				float3 out, fr;
				float pdf;
				SampleBSDF(material, -r.destination, nor, uv, dpdu, us, out, fr, pdf);
				if (!(IsBlack(fr) || pdf == 0)) {
					Intersection lightIsect;
					Ray lightRay(pos, out, r.medium, kernel_Resource.kernel_epsilon);
					if (Intersect(lightRay, &lightIsect)) {
						float3 p = lightIsect.pos;
						float3 n = lightIsect.nor;
						float3 radiance = { 0.f, 0.f, 0.f };
						if (lightIsect.lightIdx != -1)
							radiance = kernel_Resource.kernel_lights[lightIsect.lightIdx].Le(n, -lightRay.destination);
						if (!IsBlack(radiance)) {
							float pdfA, pdfW;
							kernel_Resource.kernel_lights[lightIsect.lightIdx].Pdf(Ray(p, -out, r.medium, kernel_Resource.kernel_epsilon), n, pdfA, pdfW);
							float choicePdf = PdfFromLightDistribution(lightIsect.lightIdx);
							float lenSquare = dot(p - pos, p - pos);
							float costheta = fabs(dot(n, lightRay.destination));
							float lPdf = pdfA * lenSquare / (costheta);
							float weight = PowerHeuristic(1, pdf, 1, lPdf * choicePdf);
							float3 tr = { 1.f, 1.f, 1.f };
							if (lightRay.medium) {
								if (lightRay.medium->type == MT_HOMOGENEOUS)
									tr = lightRay.medium->homogeneous.Tr(lightRay, uniform, rng);
								else
									tr = lightRay.medium->heterogeneous.Tr(lightRay, uniform, rng);
							}
							Ld += weight * tr * fr * radiance * fabs(dot(out, nor)) / pdf;
						}
					}
					else {
						//infinite
						if (kernel_Resource.kernel_infinite->isvalid) {
							float3 radiance = { 0.f, 0.f, 0.f };
							radiance = kernel_Resource.kernel_infinite->Le(lightRay.destination);
							float choicePdf = PdfFromLightDistribution(kernel_Resource.kernel_light_size);
							float lightPdf, pdfA;
							float3 lightNor;
							kernel_Resource.kernel_infinite->Pdf(lightRay, lightNor, pdfA, lightPdf);
							float weight = PowerHeuristic(1, pdf, 1, lightPdf * choicePdf);
							float3 tr = { 1.f, 1.f, 1.f };
							if (lightRay.medium) {
								if (lightRay.medium->type == MT_HOMOGENEOUS)
									tr = lightRay.medium->homogeneous.Tr(lightRay, uniform, rng);
								else
									tr = lightRay.medium->heterogeneous.Tr(lightRay, uniform, rng);
							}
							Ld += weight * tr * fr * radiance * fabs(dot(out, nor)) / pdf;
						}
					}
				}

				Li += beta * Ld;
			}

			float3 u = make_float3(uniform(rng), uniform(rng), uniform(rng));
			float3 out, fr;
			float pdf;

			SampleBSDF(material, -r.destination, nor, uv, dpdu, u, out, fr, pdf);
			if (IsBlack(fr))
				break;

			beta *= fr * fabs(dot(nor, out)) / pdf;
			specular = IsDelta(material.type);

			Medium* m = dot(out, nor) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_Resource.kernel_mediums[isect.mediumOutside])
				: (isect.mediumInside == -1 ? nullptr : &kernel_Resource.kernel_mediums[isect.mediumInside]);
			m = dot(-r.destination, nor) * dot(out, nor) > 0 ? r.medium : m;

			r = Ray(pos, out, m, kernel_Resource.kernel_epsilon);
		}

		if (bounces > 3) {
			float illumate = clamp(1.f - Luminance(beta), 0.f, 1.f);
			if (uniform(rng) < illumate)
				break;

			beta /= (1 - illumate);
		}
	}

	if (!IsInf(Li) && !IsNan(Li))
		kernel_Resource.kernel_color[pixel] = Li;
}
//**************************VolPath End****************************

