#include "hip/hip_runtime.h"
﻿#include "pathtracer.h"
#include "camera.h"
#include "scene.h"
#include "bvh.h"
#include ""
#include "Sampling.h"
#include <hiprand/hiprand_kernel.h>
#include <thrust/random.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime.h>
#include "BxDF.h"
#include "BSDF.h"

Camera* dev_camera;
float3* dev_image, * dev_color;
LinearBVHNode* dev_bvh_nodes;
Primitive* dev_primitives;
Material* dev_materials;
Bssrdf* dev_bssrdfs;
Medium* dev_mediums;
Area* dev_lights;
Infinite* dev_infinite;
float* dev_light_distribution;
uchar4** dev_textures;
int* texture_size;//0 1为第一张图的长宽， 2 3为第二张图的长宽，以此类推

__device__ Camera* kernel_camera;
__device__ int kernel_hdr_width, kernel_hdr_height;
__device__ float3* kernel_acc_image, * kernel_color;
__device__ LinearBVHNode* kernel_linear;
__device__ Primitive* kernel_primitives;
__device__ Material* kernel_materials;
__device__ Bssrdf* kernel_bssrdfs;
__device__ Medium* kernel_mediums;
__device__ Area* kernel_lights;
__device__ Infinite* kernel_infinite;
__device__ uchar4** kernel_textures;
__device__ int* kernel_texture_size;
__device__ float* kernel_light_distribution;
__device__ int kernel_light_size;
__device__ int kernel_light_distribution_size;
//不同场景需要不同的epsilon，不知道怎么样优雅的实现
__device__ float kernel_epsilon;

__host__ __device__ inline float3 SphericalDirection(float sinTheta, float cosTheta, float phi)
{
	return{ sinTheta * cosf(phi), sinTheta * sinf(phi),cosTheta };
}

__host__ __device__ inline float DielectricFresnel(float cosi, float cost, const float& etai, const float& etat) {

	float Rparl = (etat * cosi - etai * cost) / (etat * cosi + etai * cost);
	float Rperp = (etai * cosi - etat * cost) / (etai * cosi + etat * cost);

	return (Rparl * Rparl + Rperp * Rperp) * 0.5f;
}

__device__ inline float3 ConductFresnel(float cosi, const float3& eta, const float3& k) {
	float3 tmp = (eta * eta + k * k) * cosi * cosi;
	float3 Rparl2 = (tmp - eta * cosi * 2.f + 1.f) /
		(tmp + eta * cosi * 2.f + 1.f);
	float3 tmp_f = (eta * eta + k * k);
	float3 Rperp2 = (tmp_f - eta * cosi * 2.f + cosi * cosi) /
		(tmp_f + eta * cosi * 2.f + cosi * cosi);
	return (Rparl2 + Rperp2) * 0.5f;
}

__device__ inline float GGX_D(float3& wh, float3& normal, float3 dpdu, float alphaU, float alphaV) {
	float costheta = dot(wh, normal);
	if (costheta <= 0.f) return 0.f;
	costheta = clamp(costheta, 0.f, 1.f);
	float cos2Theta = costheta * costheta;
	float sin2Theta = 1.f - cos2Theta;
	float cos4Theta = cos2Theta * cos2Theta;
	float tantheta2 = sin2Theta / cos2Theta;

	float3 uu = dpdu;
	float3 dir = normalize(wh - costheta * normal);
	float cosphi = dot(dir, uu);
	float e = tantheta2 * (cosphi * cosphi / (alphaU * alphaU) + (1.f - cosphi) / (alphaV * alphaV));
	return 1.f / (PI * alphaU * alphaV * cos4Theta * (1 + e) * (1 + e));
}


__device__ inline float Lambda(float3& w, float3& normal, float3& wh, float3 dpdu, float alphaU, float alphaV) {
	float wdn = dot(w, normal);
	if (wdn * dot(w, wh) < 0.f)	return 0.f;
	float sinTheta = sqrtf(clamp(1.f - wdn * wdn, 0.f, 1.f));
	float tanTheta = sinTheta / wdn;
	if (isinf(tanTheta)) return 0.f;

	float3 uu = dpdu;
	float3 dir = normalize(w - wdn * normal);
	float cosPhi = dot(dir, uu);
	float cos2Phi = cosPhi * cosPhi;
	float sin2Phi = 1.f - cos2Phi;
	float alpha2 = cos2Phi * (alphaU * alphaU) + sin2Phi * (alphaV * alphaV);
	float alpha2Tan2Theta = alpha2 * tanTheta * tanTheta;
	return (-1.f + sqrtf(1 + alpha2Tan2Theta)) / 2;
}

__device__ inline float G1(float3& w, float3& normal, float3& wh, float3 dpdu, float alphaU, float alphaV) {
	return 1.f / 1.f + Lambda(w, normal, wh, dpdu, alphaU, alphaV);
}

__device__ inline float SmithG(float3& w, float3& normal, float3& wh, float3 dpdu, float alphaU, float alphaV) {
	float wdn = dot(w, normal);
	if (wdn * dot(w, wh) < 0.f)	return 0.f;
	float sinTheta = sqrtf(clamp(1.f - wdn * wdn, 0.f, 1.f));
	float tanTheta = sinTheta / wdn;
	if (isinf(tanTheta)) return 0.f;

	float3 uu = dpdu;
	float3 dir = normalize(w - wdn * normal);
	float cosPhi = dot(dir, uu);
	float cos2Phi = cosPhi * cosPhi;
	float sin2Phi = 1.f - cos2Phi;
	float alpha2 = cos2Phi * (alphaU * alphaU) + sin2Phi * (alphaV * alphaV);
	float alpha2Tan2Theta = alpha2 * tanTheta * tanTheta;
	return 2.f / (1.f + sqrtf(1 + alpha2Tan2Theta));
}

__device__ inline float GGX_G(float3& wo, float3& wi, float3& normal, float3& wh, float3 dpdu, float alphaU, float alphaV) {
	return SmithG(wo, normal, wh, dpdu, alphaU, alphaV) * SmithG(wi, normal, wh, dpdu, alphaU, alphaV);
}

__device__ inline float3 SampleGGX(float alphaU, float alphaV, float u1, float u2) {
	if (alphaU == alphaV) {
		float costheta = sqrtf((1.f - u1) / (u1 * (alphaU * alphaV - 1.f) + 1.f));
		float sintheta = sqrtf(1.f - costheta * costheta);
		float phi = TWOPI * u2;

		return SphericalDirection(sintheta, costheta, phi);
	}
	else {
		float phi = atan(alphaV / alphaU * tan(TWOPI * u2 + .5f * PI));
		if (u2 > .5f) phi += PI;
		float sinphi = sin(phi), cosphi = cos(phi);
		const float alphaU2 = alphaU * alphaU, alphaV2 = alphaV * alphaV;
		const float alpha2 = 1.0f / (cosphi * cosphi / alphaU2 + sinphi * sinphi / alphaU2);
		float tanTheta2 = alpha2 * u1 / (1.f - u1);
		float costheta = 1 / sqrtf(1 + tanTheta2);
		float sintheta = sqrtf(fmax((float)0., (float)1. - costheta * costheta));

		return SphericalDirection(sintheta, costheta, phi);
	}
}

__host__ __device__ inline float3 Reflect(float3 in, float3 nor) {
	return 2.f * dot(in, nor) * nor - in;
}

__host__ __device__ inline float3 Refract(float3 in, float3 nor, float etai, float etat) {
	float cosi = dot(in, nor);
	bool enter = cosi > 0;
	if (!enter) {
		float t = etai;
		etai = etat;
		etat = t;
	}

	float eta = etai / etat;
	float sini2 = 1.f - cosi * cosi;
	float sint2 = sini2 * eta * eta;
	float cost = sqrtf(1.f - sint2);
	return normalize((nor * cosi - in) * eta + (enter ? -cost : cost) * nor);
}

__device__ inline float3 SchlickFresnel(float3 specular, float cosTheta) {
	auto pow5 = [](float v) { return (v * v) * (v * v) * v; };
	float3 Rs = specular;
	return Rs + pow5(1.f - cosTheta) * (1 - Rs);
}

__device__ inline float PowerHeuristic(int nf, float fPdf, int ng, float gPdf) {
	float f = nf * fPdf, g = ng * gPdf;
	return (f * f) / (f * f + g * g);
}

//当光源多的时候可以使用二分法加速
__device__ int LookUpLightDistribution(float u, float& pdf) {
	for (int i = 0; i < kernel_light_distribution_size; ++i) {
		float s = kernel_light_distribution[i];
		float e = kernel_light_distribution[i + 1];
		if (u >= s && u <= e) {
			pdf = e - s;
			return i;
		}
	}
}

__device__ inline float PdfFromLightDistribution(int idx) {
	return kernel_light_distribution[idx + 1] - kernel_light_distribution[idx];
}

__device__ inline void GammaCorrection(float3& in) {
	float one_over_gamma = 1.f / 2.2f;
	float exposure = 1.41421356f;

	//pow(x,y) 的内部实现是expf(y*log(x)) 所以x需要大于0
	in = fmaxf(in, make_float3(1e-5, 1e-5, 1e-5));

	in.x = __powf(in.x * exposure, one_over_gamma);
	in.y = __powf(in.y * exposure, one_over_gamma);
	in.z = __powf(in.z * exposure, one_over_gamma);
}

__device__ inline void FilmicTonemapping(float3& in) {
	float3 c = in - make_float3(0.004f, 0.004f, 0.004f);
	c = fmaxf(make_float3(0, 0, 0), c);
	c = (c * (6.2f * c + 0.5f)) / (c * (6.2f * c + 1.7f) + 0.06f);
	in = c;
}

__device__ inline float Luminance(const float3& c) {
	return dot(c, { 0.212671f, 0.715160f, 0.072169f });
}

__device__ inline bool SameHemiSphere(float3& in, float3& out, float3& nor) {
	return dot(in, nor) * dot(out, nor) > 0 ? true : false;
}

__device__ bool Intersect(Ray& ray, Intersection* isect) {
	int stack[64];
	int* stack_top = stack;
	int* stack_bottom = stack;

	bool ret = false;
	int node_idx = 0;
	do {
		LinearBVHNode node = kernel_linear[node_idx];
		bool intersect = node.bbox.Intersect(ray);
		if (intersect) {
			if (!node.is_leaf) {
				*stack_top++ = node.second_child_offset;
				*stack_top++ = node_idx + 1;
			}
			else {
				for (int i = node.start; i <= node.end; ++i) {
					Primitive prim = kernel_primitives[i];

					if (prim.type == GT_TRIANGLE) {
						if (prim.triangle.Intersect(ray, isect))
							ret = true;
					}
					else if (prim.type == GT_LINES) {
						if (prim.line.Intersect(ray, isect))
							ret = true;
					}
					else if (prim.type == GT_SPHERE) {
						if (prim.sphere.Intersect(ray, isect))
							ret = true;
					}
				}
			}
		}

		if (stack_top == stack_bottom)
			break;
		node_idx = *--stack_top;
	} while (true);

	return ret;
}

__device__ bool IntersectPrimitive(Ray& ray) {
	int stack[64];
	int* stack_top = stack;
	int* stack_bottom = stack;

	int node_idx = 0;
	do {
		LinearBVHNode node = kernel_linear[node_idx];
		bool intersect = node.bbox.Intersect(ray);
		if (intersect) {
			if (!node.is_leaf) {
				*stack_top++ = node.second_child_offset;
				*stack_top++ = node_idx + 1;
			}
			else {
				for (int i = node.start; i <= node.end; ++i) {
					Primitive prim = kernel_primitives[i];
					if (prim.type == GT_TRIANGLE) {
						if (prim.triangle.Intersect(ray, nullptr))
							return true;
					}
					else if (prim.type == GT_LINES) {
						if (prim.line.Intersect(ray, nullptr))
							return true;
					}
					else if (prim.type == GT_SPHERE) {
						if (prim.sphere.Intersect(ray, nullptr))
							return true;
					}
				}
			}
		}

		if (stack_top == stack_bottom)
			break;
		node_idx = *--stack_top;
	} while (true);

	return false;
}

__device__ float3 Tr(Ray& ray, thrust::uniform_real_distribution<float>& uniform, thrust::default_random_engine& rng) {
	float3 tr = make_float3(1, 1, 1);
	float tmax = ray.tmax;
	while (true) {
		Intersection isect;
		bool invisible = Intersect(ray, &isect);
		if (invisible && isect.matIdx != -1)
			return{ 0, 0, 0 };

		if (ray.medium) {
			if (ray.medium->type == MT_HOMOGENEOUS)
				tr *= ray.medium->homogeneous.Tr(ray, uniform, rng);
			else
				tr *= ray.medium->heterogeneous.Tr(ray, uniform, rng);
		}

		if (!invisible) break;
		Medium* m = dot(ray.destination, isect.n) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_mediums[isect.mediumOutside])
			: (isect.mediumInside == -1 ? nullptr : &kernel_mediums[isect.mediumInside]);
		tmax -= ray.tmax;
		ray = Ray(ray(ray.tmax), ray.destination, m, kernel_epsilon, tmax);
	}

	return tr;
}

__device__ inline float4 getTexel(Material material, int w, int h, int2 uv) {
	float inv = 1.f / 255.f;

	int x = uv.x, y = uv.y;
	float rx = x - (x / w) * w;
	float ry = y - (y / h) * h;
	x = (rx < 0) ? rx + w : rx;
	y = (ry < 0) ? ry + h : ry;
	if (x < 0) x = 0;
	if (x > w - 1) x = w - 1;
	if (y < 0) y = 0;
	if (y > h - 1) y = h - 1;

	uchar4 c = kernel_textures[material.textureIdx][y * w + x];
	return make_float4(c.x * inv, c.y * inv, c.z * inv, c.w * inv);
}

__device__ inline float4 GetTexel(Material material, float2 uv) {
	if (material.textureIdx == -1)
		return make_float4(material.diffuse, 1.f);

	int w = kernel_texture_size[material.textureIdx * 2];
	int h = kernel_texture_size[material.textureIdx * 2 + 1];
	float xx = w * uv.x;
	float yy = h * uv.y;
	int x = floor(xx);
	int y = floor(yy);
	float dx = fabs(xx - x);
	float dy = fabs(yy - y);
	float4 c00 = getTexel(material, w, h, make_int2(x, y));
	float4 c10 = getTexel(material, w, h, make_int2(x + 1, y));
	float4 c01 = getTexel(material, w, h, make_int2(x, y + 1));
	float4 c11 = getTexel(material, w, h, make_int2(x + 1, y + 1));
	return (1 - dy) * ((1 - dx) * c00 + dx * c10)
		+ dy * ((1 - dx) * c01 + dx * c11);
}

//**************************bssrdf*****************
__device__ float3 SingleScatter(Intersection* isect, float3 in, thrust::uniform_real_distribution<float>& uniform, thrust::default_random_engine& rng) {
	float3 pos = isect->pos;
	float3 nor = isect->n;
	float coso = fabs(dot(in, nor));
	Bssrdf bssrdf = kernel_bssrdfs[isect->bssrdf];
	float eta = bssrdf.eta;
	float sino2 = 1.f - coso * coso;
	float cosi = sqrtf(1.f - sino2 / (eta * eta));
	float fresnel = 1.f - DielectricFresnel(coso, cosi, 1.f, eta);
	float sigmaTr = Luminance(bssrdf.GetSigmaTr());
	float3 sigmaS = bssrdf.GetSigmaS();
	float3 sigmaT = bssrdf.GetSigmaT();
	float3 rdir = Reflect(in, nor);
	float3 tdir = Refract(in, nor, 1.f, eta);
	float3 L = { 0, 0, 0 };
	Intersection rIsect;
	if (Intersect(Ray(pos, rdir, nullptr, kernel_epsilon), &rIsect)) {
		if (rIsect.lightIdx != -1) {
			L += (1.f - fresnel) * kernel_lights[rIsect.lightIdx].Le(rIsect.n, -rdir);
		}
	}
	Intersection tIsect;
	Intersect(Ray(pos, tdir, nullptr, kernel_hdr_height), &tIsect);
	float len = length(tIsect.pos - pos);
	int samples = 1;
	for (int i = 0; i < samples; ++i) {
		float d = Exponential(uniform(rng), sigmaTr);
		if (d > len) continue;
		float3 pSample = pos + tdir * d;
		float pdf = ExponentialPdf(d, sigmaTr);
		float choicePdf;
		float u = uniform(rng);
		int idx = LookUpLightDistribution(u, choicePdf);
		Area light = kernel_lights[idx];
		float lightPdf;
		Ray shadowRay;
		float3 radiance, lightNor;
		float2 u1 = make_float2(uniform(rng), uniform(rng));
		light.SampleLight(pSample, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);
		if (IsBlack(radiance))
			continue;

		float tmax = shadowRay.tmax;
		Intersection wiIsect;
		if (Intersect(shadowRay, &wiIsect)) {
			if (wiIsect.bssrdf == isect->bssrdf) {
				float3 wiPos = wiIsect.pos;
				float3 wiNor = wiIsect.n;
				shadowRay.tmin += shadowRay.tmax;
				shadowRay.tmax = tmax;
				if (!IntersectPrimitive(shadowRay)) {
					float p = bssrdf.GetPhase();
					float cosi = fabs(dot(wiNor, shadowRay.destination));
					float sini2 = 1.f - cosi * cosi;
					float coso = sqrtf(1.f - sini2 / (eta * eta));
					float fresnelI = 1.f - DielectricFresnel(cosi, coso, 1.f, eta);
					float G = fabs(dot(wiNor, tdir)) / cosi;
					float3 sigmaTC = sigmaT * (1.f + G);
					float di = length(wiPos - pSample);
					float et = 1.f / eta;
					float diPrime = di * fabs(dot(shadowRay.destination, wiNor)) /
						sqrt(1.f - et * et * (1.f - cosi * cosi));
					L += (fresnel * fresnelI * p * sigmaS / sigmaTC) *
						Exp(-diPrime * sigmaT) *
						Exp(-d * sigmaT) * radiance / (lightPdf * choicePdf * pdf);
				}
			}
		}
	}

	L /= samples;
	return L;
}

__device__ float3 MultipleScatter(Intersection* isect, float3 in, thrust::uniform_real_distribution<float>& uniform, thrust::default_random_engine& rng) {
	float3 pos = isect->pos;
	float3 nor = isect->n;
	float coso = fabs(dot(in, nor));
	Bssrdf bssrdf = kernel_bssrdfs[isect->bssrdf];
	float eta = bssrdf.eta;
	float sino2 = 1.f - coso * coso;
	float cosi = sqrtf(1.f - sino2 / (eta * eta));
	float fresnel = 1.f - DielectricFresnel(coso, cosi, 1.f, eta);
	float sigmaTr = Luminance(bssrdf.GetSigmaTr());
	float skipRatio = 0.01f;
	float rMax = sqrt(log(skipRatio) / -sigmaTr);
	float3 L = { 0, 0, 0 };
	int samples = 1;
	for (int i = 0; i < samples; ++i) {
		Ray probeRay;
		float pdf;
		float2 u = make_float2(uniform(rng), uniform(rng));
		bssrdf.SampleProbeRay(pos, nor, u, sigmaTr, rMax, probeRay, pdf);
		probeRay.tmin = kernel_epsilon;

		Intersection probeIsect;
		if (Intersect(probeRay, &probeIsect)) {
			if (isect->bssrdf == probeIsect.bssrdf) {
				float3 probePos = probeIsect.pos;
				float3 probeNor = probeIsect.n;
				float3 rd = bssrdf.Rd(dot(probePos - pos, probePos - pos));
				float choicePdf;
				float u = uniform(rng);
				int idx = LookUpLightDistribution(u, choicePdf);
				Area light = kernel_lights[idx];
				float lightPdf;
				float2 u1 = make_float2(uniform(rng), uniform(rng));
				float3 radiance, lightNor;
				Ray shadowRay;
				light.SampleLight(probePos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);
				if (!IsBlack(radiance) && !IntersectPrimitive(shadowRay)) {
					float cosi = fabs(dot(shadowRay.destination, probeNor));
					float sini2 = 1.f - cosi * cosi;
					float cost = sqrtf(1.f - sini2 / (eta * eta));
					float3 irradiance = radiance * cosi / (lightPdf * choicePdf);
					float fresnelI = 1.f - DielectricFresnel(cosi, cost, 1.f, eta);
					pdf *= fabs(dot(probeRay.destination, probeNor));
					L += (ONE_OVER_PI * fresnel * fresnelI * rd * irradiance) / pdf;
				}
			}
		}

		L /= samples;
		return L;
	}
}
//**************************bssrdf end*************

//**************************BSDF Sampling**************************
__device__ void SampleBSDF(Material material, float3 in, float3 nor, float2 uv, float3 dpdu, float3 u, float3& out, float3& fr, float& pdf, TransportMode mode = TransportMode::Radiance) {
	switch (material.type) {
	case MT_LAMBERTIAN: {
		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;

		out = CosineSampleHemiSphere(u.x, u.y, n, pdf);
		float3 uu = dpdu, ww;
		ww = cross(uu, n);
		out = ToWorld(out, uu, ww, n);
		fr = make_float3(GetTexel(material, uv)) * ONE_OVER_PI;
		break;
	}

	case MT_MIRROR:
		out = Reflect(in, nor);
		fr = material.specular / fabs(dot(out, nor));
		pdf = 1.f;
		break;

	case MT_DIELECTRIC: {
		// translucent
		float3 wi = -in;
		float3 normal = nor;

		float ei = material.outsideIOR, et = material.insideIOR;
		float cosi = dot(wi, normal);
		if (cosi > 0)
			swap(et, ei);
		float eta = ei / et;

		float cost;
		float sint2 = eta * eta * (1.f - cosi * cosi);
		cost = sqrtf(1.f - sint2 < 0.f ? 0.f : 1.f - sint2);
		float3 rdir = Reflect(-wi, normal);
		float3 tdir = Refract(in, nor, material.outsideIOR, material.insideIOR);
		if (sint2 > 1.f) {//total reflection
			out = rdir;
			fr = material.specular / fabs(dot(out, normal));
			pdf = 1.f;
			return;
		}

		float fresnel = DielectricFresnel(fabs(cost), fabs(cosi), et, ei);
		if (u.x > fresnel) {
			//refract
			out = tdir;
			fr = material.specular / fabs(dot(out, normal)) * (1.f - fresnel);
			if (mode == TransportMode::Radiance)
				fr *= eta * eta;
			pdf = 1.f - fresnel;
		}
		else {
			//reflect
			out = rdir;
			fr = material.specular / fabs(dot(out, normal)) * fresnel;
			pdf = fresnel;
		}
		break;
	}

	case MT_ROUGHCONDUCTOR: {
		// Metal
		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;

		float3 wh = SampleGGX(material.alphaU, material.alphaV, u.x, u.y);
		float3 uu = dpdu, ww;
		ww = cross(uu, n);
		wh = ToWorld(wh, uu, ww, n);
		out = Reflect(in, wh);
		if (!SameHemiSphere(in, out, nor)) {
			fr = { 0, 0, 0 };
			pdf = 0.f;
			return;
		}

		float cosi = dot(out, wh);
		float3 F = ConductFresnel(fabs(cosi), material.eta, material.k);
		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);

		fr = material.specular * F * D * G /
			(4.f * fabs(dot(in, n)) * fabs(dot(out, n)));
		pdf = D * fabs(dot(wh, n)) / (4.f * fabs(dot(in, wh)));
		break;
	}

	case MT_SUBSTRATE: {
		// sampleVisibleArea = false
		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;
		if (u.x < 0.5) {
			u.x = min(2.f * u.x, 0x1.fffffep-1);
			//float ux = u.x * 2.f;
			out = CosineSampleHemiSphere(u.x, u.y, n, pdf);
			float3 uu = dpdu, ww;
			ww = cross(uu, n);
			out = ToWorld(out, uu, ww, n);
		}
		else {
			u.x = min(2 * (u.x - 0.5f), 0x1.fffffep-1);
			//float ux = (u.x - 0.5f) * 2.f;
			float3 wh = SampleGGX(material.alphaU, material.alphaV, u.x, u.y);
			float3 uu = dpdu, ww;
			ww = cross(uu, n);
			wh = ToWorld(wh, uu, ww, n);
			out = Reflect(in, wh);
		}
		if (!SameHemiSphere(in, out, n)) {
			fr = { 0.f, 0.f, 0.f };
			pdf = 0.f;
			return;
		}

		float cosThetaWo = fabs(dot(in, n));
		float cosThetaWi = fabs(dot(out, n));
		float3 Rd = make_float3(GetTexel(material, uv));
		float3 Rs = material.specular;

		auto pow5 = [](float v) { return (v * v) * (v * v) * v; };
		float3 diffuse = (28.f / (23.f * PI)) * Rd * (make_float3(1.f, 1.f, 1.f) - Rs) *
			(1 - pow5(1 - .5f * cosThetaWi)) *
			(1 - pow5(1 - .5f * cosThetaWo));

		float3 wh = normalize(in + out);
		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		float3 specular = D /
			(4.f * fabs(dot(out, wh)) * Max(cosThetaWi, cosThetaWo)) *
			SchlickFresnel(Rs, dot(out, wh));

		//float G = G1(in, n, wh, dpdu, material.alphaU, material.alphaV);

		fr = diffuse + specular;
		pdf = 0.5f * (fabs(dot(out, n)) * ONE_OVER_PI + D * fabs(dot(wh, n)) / (4.f * dot(in, wh)));

		break;
	}

	case MT_ROUGHDIELECTRIC: {
		float3 wo = -in;
		float3 n = nor;
		float3 wh = SampleGGX(material.alphaU, material.alphaV, u.x, u.y);
		float3 uu = dpdu, ww;
		ww = cross(uu, n);
		wh = ToWorld(wh, uu, ww, n);

		float ei = material.outsideIOR, et = material.insideIOR;
		float cosi = dot(wo, n);
		if (cosi > 0)
			swap(et, ei);
		float eta = ei / et;

		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);

		float cosThetaI = dot(wo, wh);
		float sin2ThetaI = fmaxf(0.f, 1.f - cosThetaI * cosThetaI);
		float sin2ThetaT = eta * eta * sin2ThetaI;
		float cosThetaT = sqrtf(1.f - sin2ThetaT < 0.f ? 0.f : 1.f - sin2ThetaT);

		float3 rdir = Reflect(-wo, wh);
		float3 tdir = normalize((wo - wh * cosThetaI) * eta + (cosi < 0 ? -cosThetaT : cosThetaT) * wh);

		if (sin2ThetaT > 1.f) {
			//total reflection
			out = rdir;
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			fr = material.specular * D * G / (4.f * fabs(dot(in, n)) * fabs(dot(out, n)));
			pdf = D * fabs(dot(wh, n)) / (4.f * fabs(dot(wh, in)));
			return;
		}

		float fresnel = DielectricFresnel(fabs(cosThetaI), fabs(cosThetaT), ei, et);
		if (u.z > fresnel) {
			//refract
			out = tdir;
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			float c = et * dot(out, wh) + ei * dot(in, wh);
			fr = material.specular * ei * ei * D * G * (1.f - fresnel) * fabs(dot(in, wh)) * fabs(dot(out, wh)) /
				(fabs(dot(out, n)) * fabs(dot(in, n)) * c * c);
			if (mode == TransportMode::Radiance)
				fr *= (1.f / (eta * eta));

			pdf = (1.f - fresnel) * D * fabs(dot(wh, n)) * et * et * fabs(dot(out, wh)) / (c * c);
		}
		else {
			//reflect
			out = rdir;
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			fr = material.specular * fresnel * D * G / (4.f * fabs(dot(in, n)) * fabs(dot(out, n)));
			pdf = D * fabs(dot(wh, n)) / (4.f * fabs(dot(wh, in))) * fresnel;
		}
		break;
	}
	}
}

__device__ void Fr(Material material, float3 in, float3 out, float3 nor, float2 uv, float3 dpdu, float3& fr, float& pdf, TransportMode mode = TransportMode::Radiance) {
	switch (material.type) {
	case MT_LAMBERTIAN:
		if (!SameHemiSphere(in, out, nor)) {
			fr = make_float3(0.f, 0.f, 0.f);
			pdf = 0.f;
			return;
		}

		fr = make_float3(GetTexel(material, uv)) * ONE_OVER_PI;
		pdf = fabs(dot(out, nor)) * ONE_OVER_PI;
		break;

	case MT_MIRROR:
		fr = make_float3(0.f, 0.f, 0.f);
		pdf = 0.f;
		break;

	case MT_DIELECTRIC:
		fr = make_float3(0.f, 0.f, 0.f);
		pdf = 0.f;
		break;

	case MT_ROUGHCONDUCTOR: {
		if (!SameHemiSphere(in, out, nor)) {
			fr = { 0, 0, 0 };
			pdf = 0;
			return;
		}
		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;

		float3 wh = normalize(in + out);
		float cosi = dot(out, wh);
		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
		float3 F = ConductFresnel(fabs(cosi), material.eta, material.k);
		fr = material.specular * F * D * G /
			(4.f * fabs(dot(in, n)) * fabs(dot(out, n)));
		pdf = D * fabs(dot(wh, n)) / (4.f * fabs(dot(in, wh)));
		break;
	}

	case MT_SUBSTRATE: {
		if (!SameHemiSphere(in, out, nor)) {
			fr = { 0, 0, 0 };
			pdf = 0;
			return;
		}

		float3 n = nor;
		if (dot(nor, in) < 0)
			n = -n;

		float cosThetaWo = fabs(dot(in, n));
		float cosThetaWi = fabs(dot(out, n));
		float3 Rd = make_float3(GetTexel(material, uv));
		float3 Rs = material.specular;

		auto pow5 = [](float v) { return (v * v) * (v * v) * v; };
		float3 diffuse = (28.f / (23.f * PI)) * Rd * (make_float3(1.f, 1.f, 1.f) - Rs) *
			(1 - pow5(1 - .5f * cosThetaWi)) *
			(1 - pow5(1 - .5f * cosThetaWo));

		float3 wh = normalize(in + out);
		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);
		float3 specular = D /
			(4.f * fabs(dot(out, wh)) * Max(cosThetaWi, cosThetaWo)) *
			SchlickFresnel(Rs, dot(out, wh));

		fr = diffuse + specular;
		pdf = 0.5f * (fabs(dot(out, n)) * ONE_OVER_PI + D * fabs(dot(wh, n)) / (4.f * dot(in, wh)));

		break;
	}

	case MT_ROUGHDIELECTRIC: {
		float3 wo = -in;
		float3 n = nor;
		bool reflect = dot(in, n) * dot(out, n) > 0;

		float ei = material.outsideIOR, et = material.insideIOR;
		float cosi = dot(wo, n);
		if (cosi > 0)
			swap(et, ei);

		float3 wh = normalize(-(ei * in + et * out));
		float eta = ei / et;

		float D = GGX_D(wh, n, dpdu, material.alphaU, material.alphaV);

		float cosThetaI = dot(wo, wh);
		float sin2ThetaI = fmaxf(0.f, 1.f - cosThetaI * cosThetaI);
		float sin2ThetaT = eta * eta * sin2ThetaI;
		float cosThetaT = sqrtf(1.f - sin2ThetaT < 0.f ? 0.f : 1.f - sin2ThetaT);

		float fresnel = DielectricFresnel(fabs(cosThetaI), fabs(cosThetaT), ei, et);

		if (!reflect) {
			//refract
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			float c = et * dot(out, wh) + ei * dot(in, wh);
			fr = material.specular * ei * ei * D * G * (1.f - fresnel) * fabs(dot(in, wh)) * fabs(dot(out, wh)) /
				(fabs(dot(out, n)) * fabs(dot(in, n)) * c * c);
			if (mode == TransportMode::Radiance)
				fr *= (1.f / (eta * eta));
			pdf = (1.f - fresnel) * D * fabs(dot(wh, n)) * et * et * fabs(dot(out, wh)) / (c * c);
		}
		else {
			float G = GGX_G(in, out, n, wh, dpdu, material.alphaU, material.alphaV);
			fr = material.specular * fresnel * D * G / (4.f * fabs(dot(in, n)) * fabs(dot(out, n)));
			pdf = fresnel * D * fabs(dot(wh, n)) / (4.f * fabs(dot(wh, in)));

		}
		break;
	}
	}
}
//**************************BSDF End*******************************

//**************************AO Integrator**************************
__global__ void Ao(int iter, float maxDist) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	//init seed
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	//start
	float offsetx = uniform(rng) - 0.5f;
	float offsety = uniform(rng) - 0.5f;
	float unuse;
	float2 aperture = UniformSampleDisk(uniform(rng), uniform(rng), unuse);//for dof
	Ray ray = kernel_camera->GeneratePrimaryRay(x + offsetx, y + offsety, aperture);
	ray.tmin = kernel_epsilon;

	float3 L = { 0.f, 0.f, 0.f };
	Intersection isect;
	bool intersect = Intersect(ray, &isect);
	if (!intersect) {
		kernel_color[pixel] += { 1, 1, 1 };
		return;
	}

	float3 pos = isect.pos;
	float3 n = isect.n;
	float pdf = 0.f;
	if (dot(-ray.destination, n) < 0.f)
		n = -n;
	float3 wi = CosineSampleHemiSphere(uniform(rng), uniform(rng), n, pdf);
	float3 s = isect.dpdu;
	float3 t = cross(n, s);
	wi = ToWorld(wi, s, t, n);

	float cosine = dot(wi, n);
	Ray r(pos, wi, nullptr, kernel_epsilon, maxDist);
	intersect = IntersectPrimitive(r);
	if (!intersect) {
		float v = cosine / pdf;
		L += make_float3(v, v, v);
	}

	if (!IsNan(L))
		kernel_color[pixel] = L;
}
//**************************AO End*********************************

//**************************Path Integrator************************
__global__ void Path(int iter, int maxDepth) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	//init seed
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	//start
	float offsetx = uniform(rng) - 0.5f;
	float offsety = uniform(rng) - 0.5f;
	float unuse;
	float2 aperture = UniformSampleDisk(uniform(rng), uniform(rng), unuse);//for dof
	Ray ray = kernel_camera->GeneratePrimaryRay(x + offsetx, y + offsety, aperture);
	ray.tmin = kernel_epsilon;

	float3 Li = make_float3(0.f, 0.f, 0.f);
	float3 beta = make_float3(1.f, 1.f, 1.f);
	Ray r = ray;
	Intersection isect;
	bool specular = false;
	for (int bounces = 0; bounces < maxDepth; ++bounces) {
		if (!Intersect(r, &isect)) {
			if ((bounces == 0 || specular) && kernel_infinite->isvalid)
				Li += beta * kernel_infinite->Le(r.destination);
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.n;
		float2 uv = isect.uv;
		float3 dpdu = isect.dpdu;
		Material material = kernel_materials[isect.matIdx];

		if (bounces == 0 || specular) {
			if (isect.lightIdx != -1) {
				Li += beta * kernel_lights[isect.lightIdx].Le(nor, -r.destination);
				break;
			}
		}

		//direct light with multiple importance sampling
		if (!IsDelta(material.type)) {
			float3 Ld = make_float3(0.f, 0.f, 0.f);
			float choicePdf;
			int idx = LookUpLightDistribution(uniform(rng), choicePdf);
			float2 u1 = make_float2(uniform(rng), uniform(rng));
			float3 radiance, lightNor;
			Ray shadowRay;
			float lightPdf;
			if (idx != kernel_light_size)
				kernel_lights[idx].SampleLight(pos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);
			else
				kernel_infinite->SampleLight(pos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);
			shadowRay.medium = r.medium;

			// Compute effect of visibility for light source sample
			if (!IsBlack(radiance) && !IntersectPrimitive(shadowRay)) {
				float3 fr;
				float samplePdf;

				Fr(material, -r.destination, shadowRay.destination, nor, uv, dpdu, fr, samplePdf);

				float weight = PowerHeuristic(1, lightPdf * choicePdf, 1, samplePdf);
				Ld += weight * fr * radiance * fabs(dot(nor, shadowRay.destination)) / (lightPdf * choicePdf);
			}

			float3 us = make_float3(uniform(rng), uniform(rng), uniform(rng));
			float3 out, fr;
			float pdf;
			SampleBSDF(material, -r.destination, nor, uv, dpdu, us, out, fr, pdf);

			// Add light's contribution to reflected radiance
			if (!IsBlack(fr) && pdf != 0) {
				Intersection lightIsect;
				Ray lightRay(pos, out, r.medium, kernel_epsilon);
				if (Intersect(lightRay, &lightIsect)) {
					float3 p = lightIsect.pos;
					float3 n = lightIsect.n;
					float3 radiance = { 0.f, 0.f, 0.f };
					if (lightIsect.lightIdx != -1)
						radiance = kernel_lights[lightIsect.lightIdx].Le(n, -lightRay.destination);
					if (!IsBlack(radiance)) {
						float pdfA, pdfW;
						kernel_lights[lightIsect.lightIdx].Pdf(Ray(p, -out, r.medium, kernel_epsilon), n, pdfA, pdfW);
						float choicePdf = PdfFromLightDistribution(lightIsect.lightIdx);
						float lenSquare = dot(p - pos, p - pos);
						float costheta = fabs(dot(n, lightRay.destination));
						float lPdf = pdfA * lenSquare / (costheta);
						float weight = PowerHeuristic(1, pdf, 1, lPdf * choicePdf);

						Ld += weight * fr * radiance * fabs(dot(out, nor)) / pdf;
					}
				}
				else {
					//infinite
					if (kernel_infinite->isvalid) {
						float3 radiance = { 0.f, 0.f, 0.f };
						radiance = kernel_infinite->Le(lightRay.destination);
						float choicePdf = PdfFromLightDistribution(kernel_light_size);
						float lightPdf, pdfA;
						float3 lightNor;
						kernel_infinite->Pdf(lightRay, lightNor, pdfA, lightPdf);
						float weight = PowerHeuristic(1, pdf, 1, lightPdf * choicePdf);

						Ld += weight * fr * radiance * fabs(dot(out, nor)) / pdf;
					}
				}
			}

			Li += beta * Ld;
		}

		// Sample BSDF to get new path direction
		float3 out, fr;
		float pdf;

		SampleBSDF(material, -r.destination, nor, uv, dpdu, make_float3(uniform(rng), uniform(rng), uniform(rng)), out, fr, pdf);
		if (IsBlack(fr))
			break;

		beta *= fr * fabs(dot(nor, out)) / pdf;
		specular = IsDelta(material.type);

		r = Ray(pos, out, nullptr, kernel_epsilon);

		if (bounces > 3) {
			float illumate = clamp(1.f - Luminance(beta), 0.f, 1.f);
			if (uniform(rng) < illumate)
				break;

			beta /= (1 - illumate);
		}
	}

	if (!IsInf(Li) && !IsNan(Li))
		kernel_color[pixel] = Li;
}
//**************************Path End*******************************

//**************************VolPath Integrator*********************
__global__ void Volpath(int iter, int maxDepth) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	//init seed
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	//start
	float offsetx = uniform(rng) - 0.5f;
	float offsety = uniform(rng) - 0.5f;
	float unuse;
	float2 aperture = UniformSampleDisk(uniform(rng), uniform(rng), unuse);//for dof
	Ray ray = kernel_camera->GeneratePrimaryRay(x + offsetx, y + offsety, aperture);
	ray.tmin = kernel_epsilon;
	ray.medium = kernel_camera->medium == -1 ? nullptr : &kernel_mediums[kernel_camera->medium];

	float3 Li = make_float3(0.f, 0.f, 0.f);
	float3 beta = make_float3(1.f, 1.f, 1.f);
	Ray r = ray;
	Intersection isect;
	bool specular = false;
	for (int bounces = 0; bounces < maxDepth; ++bounces) {
		if (!Intersect(r, &isect)) {
			if ((bounces == 0 || specular) && kernel_infinite->isvalid)
				Li += beta * kernel_infinite->Le(r.destination);
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.n;
		float2 uv = isect.uv;
		float3 dpdu = isect.dpdu;

		float sampledDist;
		bool sampledMedium = false;
		if (r.medium) {
			if (r.medium->type == MT_HOMOGENEOUS)
				beta *= r.medium->homogeneous.Sample(r, uniform, rng, sampledDist, sampledMedium);
			else
				beta *= r.medium->heterogeneous.Sample(r, uniform, rng, sampledDist, sampledMedium);
		}
		if (IsBlack(beta)) break;
		if (sampledMedium) {
			//TODO:������Ҫ�Բ���
			bool inf = false;
			float u = uniform(rng);
			float choicePdf;
			int idx = LookUpLightDistribution(u, choicePdf);
			if (idx == kernel_light_size) inf = true;
			float3 samplePos = r(sampledDist);
			float2 u1 = make_float2(uniform(rng), uniform(rng));
			float3 radiance, lightNor;
			Ray shadowRay;
			float lightPdf;
			if (!inf)
				kernel_lights[idx].SampleLight(samplePos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);
			else
				kernel_infinite->SampleLight(samplePos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);
			shadowRay.medium = r.medium;
			float3 tr = Tr(shadowRay, uniform, rng);
			float phase, unuse;
			r.medium->Phase(-r.destination, shadowRay.destination, phase, unuse);

			if (!IsBlack(radiance))
				Li += tr * beta * phase * radiance / (lightPdf * choicePdf);

			float pdf;
			float2 phaseU = make_float2(uniform(rng), uniform(rng));
			float3 dir;
			r.medium->SamplePhase(phaseU, dir, phase, pdf);
			r = Ray(samplePos, dir, r.medium, kernel_epsilon);
			specular = false;
		}
		else {
			if (bounces == 0 || specular) {
				if (isect.lightIdx != -1) {
					float3 tr = { 1.f, 1.f, 1.f };
					if (r.medium) {
						if (r.medium->type == MT_HOMOGENEOUS)
							tr = r.medium->homogeneous.Tr(r, uniform, rng);
						else
							tr = r.medium->heterogeneous.Tr(r, uniform, rng);
					}
					Li += tr * beta * kernel_lights[isect.lightIdx].Le(nor, -r.destination);
					break;
				}
			}

			if (isect.matIdx == -1) {
				bounces--;
				Medium* m = dot(r.destination, isect.n) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_mediums[isect.mediumOutside])
					: (isect.mediumInside == -1 ? nullptr : &kernel_mediums[isect.mediumInside]);
				r = Ray(pos, r.destination, m, kernel_epsilon);

				continue;
			}

			Material material = kernel_materials[isect.matIdx];
			//direct light with multiple importance sampling
			if (!IsDelta(material.type)) {
				float3 Ld = make_float3(0.f, 0.f, 0.f);
				bool inf = false;
				float u = uniform(rng);
				float choicePdf;
				int idx = LookUpLightDistribution(u, choicePdf);
				if (idx == kernel_light_size) inf = true;
				float2 u1 = make_float2(uniform(rng), uniform(rng));
				float3 radiance, lightNor;
				Ray shadowRay;
				float lightPdf;
				if (!inf)
					kernel_lights[idx].SampleLight(pos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);
				else
					kernel_infinite->SampleLight(pos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);
				shadowRay.medium = r.medium;

				if (!IsBlack(radiance)) {
					float3 fr;
					float samplePdf;

					//Fr(material, -r.destination, shadowRay.destination, nor, uv, dpdu, uniform(rng), fr, samplePdf);
					Fr(material, -r.destination, shadowRay.destination, nor, uv, dpdu, fr, samplePdf);
					float3 tr = Tr(shadowRay, uniform, rng);

					float weight = PowerHeuristic(1, lightPdf * choicePdf, 1, samplePdf);
					Ld += weight * tr * fr * radiance * fabs(dot(nor, shadowRay.destination)) / (lightPdf * choicePdf);
				}

				float3 us = make_float3(uniform(rng), uniform(rng), uniform(rng));
				float3 out, fr;
				float pdf;
				SampleBSDF(material, -r.destination, nor, uv, dpdu, us, out, fr, pdf);
				if (!(IsBlack(fr) || pdf == 0)) {
					Intersection lightIsect;
					Ray lightRay(pos, out, r.medium, kernel_epsilon);
					if (Intersect(lightRay, &lightIsect)) {
						float3 p = lightIsect.pos;
						float3 n = lightIsect.n;
						float3 radiance = { 0.f, 0.f, 0.f };
						if (lightIsect.lightIdx != -1)
							radiance = kernel_lights[lightIsect.lightIdx].Le(n, -lightRay.destination);
						if (!IsBlack(radiance)) {
							float pdfA, pdfW;
							kernel_lights[lightIsect.lightIdx].Pdf(Ray(p, -out, r.medium, kernel_epsilon), n, pdfA, pdfW);
							float choicePdf = PdfFromLightDistribution(lightIsect.lightIdx);
							float lenSquare = dot(p - pos, p - pos);
							float costheta = fabs(dot(n, lightRay.destination));
							float lPdf = pdfA * lenSquare / (costheta);
							float weight = PowerHeuristic(1, pdf, 1, lPdf * choicePdf);
							float3 tr = { 1.f, 1.f, 1.f };
							if (lightRay.medium) {
								if (lightRay.medium->type == MT_HOMOGENEOUS)
									tr = lightRay.medium->homogeneous.Tr(lightRay, uniform, rng);
								else
									tr = lightRay.medium->heterogeneous.Tr(lightRay, uniform, rng);
							}
							Ld += weight * tr * fr * radiance * fabs(dot(out, nor)) / pdf;
						}
					}
					else {
						//infinite
						if (kernel_infinite->isvalid) {
							float3 radiance = { 0.f, 0.f, 0.f };
							radiance = kernel_infinite->Le(lightRay.destination);
							float choicePdf = PdfFromLightDistribution(kernel_light_size);
							float lightPdf, pdfA;
							float3 lightNor;
							kernel_infinite->Pdf(lightRay, lightNor, pdfA, lightPdf);
							float weight = PowerHeuristic(1, pdf, 1, lightPdf * choicePdf);
							float3 tr = { 1.f, 1.f, 1.f };
							if (lightRay.medium) {
								if (lightRay.medium->type == MT_HOMOGENEOUS)
									tr = lightRay.medium->homogeneous.Tr(lightRay, uniform, rng);
								else
									tr = lightRay.medium->heterogeneous.Tr(lightRay, uniform, rng);
							}
							Ld += weight * tr * fr * radiance * fabs(dot(out, nor)) / pdf;
						}
					}
				}

				Li += beta * Ld;
			}

			float3 u = make_float3(uniform(rng), uniform(rng), uniform(rng));
			float3 out, fr;
			float pdf;

			SampleBSDF(material, -r.destination, nor, uv, dpdu, u, out, fr, pdf);
			if (IsBlack(fr))
				break;

			beta *= fr * fabs(dot(nor, out)) / pdf;
			specular = IsDelta(material.type);

			Medium* m = dot(out, nor) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_mediums[isect.mediumOutside])
				: (isect.mediumInside == -1 ? nullptr : &kernel_mediums[isect.mediumInside]);
			m = dot(-r.destination, nor) * dot(out, nor) > 0 ? r.medium : m;

			r = Ray(pos, out, m, kernel_epsilon);
		}

		if (bounces > 3) {
			float illumate = clamp(1.f - Luminance(beta), 0.f, 1.f);
			if (uniform(rng) < illumate)
				break;

			beta /= (1 - illumate);
		}
	}

	if (!IsInf(Li) && !IsNan(Li))
		kernel_color[pixel] = Li;
}
//**************************VolPath End****************************

//**************************Lighttracing Integrator****************
__global__ void LightTracingInit() {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	kernel_color[pixel] = { 0.f, 0.f, 0.f };
}

__global__ void LightTracing(int iter, int maxDepth) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	//init seed
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	float3 beta = { 1.f, 1.f, 1.f };
	float choicePdf;
	int lightIdx = LookUpLightDistribution(uniform(rng), choicePdf);
	Area light = kernel_lights[lightIdx];
	float4 u = make_float4(uniform(rng), uniform(rng), uniform(rng), uniform(rng));
	Ray ray;
	float3 nor, radiance;
	float pdfA, pdfW;
	light.SampleLight(u, ray, nor, radiance, pdfA, pdfW, kernel_epsilon);
	ray.medium = light.medium == -1 ? nullptr : &kernel_mediums[light.medium];

	beta *= radiance * fabs(dot(ray.destination, nor)) / (pdfA * pdfW * choicePdf);

	Ray shadowRay;
	float we, cameraPdf;
	int raster;
	kernel_camera->SampleCamera(ray.origin, shadowRay, we, cameraPdf, raster, kernel_epsilon);
	shadowRay.medium = ray.medium;
	if (cameraPdf != 0.f) {
		float3 tr = Tr(shadowRay, uniform, rng);
		if (!IsBlack(tr))
			kernel_color[raster] += tr * radiance;
	}

	Intersection isect;
	for (int bounces = 0; bounces < maxDepth; ++bounces) {
		if (!Intersect(ray, &isect)) {
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.n;
		float2 uv = isect.uv;
		float sampledDist;
		bool sampledMedium = false;
		if (ray.medium) {
			if (ray.medium->type == MT_HOMOGENEOUS)
				beta *= ray.medium->homogeneous.Sample(ray, uniform, rng, sampledDist, sampledMedium);
			else
				beta *= ray.medium->heterogeneous.Sample(ray, uniform, rng, sampledDist, sampledMedium);
		}
		if (IsBlack(beta)) break;
		if (sampledMedium) {
			float3 samplePos = ray(sampledDist);
			Ray shadowRay;
			float we, cameraPdf;
			int raster;
			kernel_camera->SampleCamera(samplePos, shadowRay, we, cameraPdf, raster, kernel_epsilon);
			shadowRay.medium = ray.medium;
			float3 tr = Tr(shadowRay, uniform, rng);
			float phase, unuse;
			ray.medium->Phase(-ray.destination, shadowRay.destination, phase, unuse);

			float3 L = beta * we * tr * phase / cameraPdf;
			if (!IsInf(L) && !IsNan(L)) {
				//kernel_color[raster] += L;
				atomicAdd(&kernel_color[raster].x, L.x);
				atomicAdd(&kernel_color[raster].y, L.y);
				atomicAdd(&kernel_color[raster].z, L.z);
			}

			float pdf;
			float2 phaseU = make_float2(uniform(rng), uniform(rng));
			float3 dir;
			ray.medium->SamplePhase(phaseU, dir, phase, pdf);
			ray = Ray(samplePos, dir, ray.medium, kernel_epsilon);
		}
		else {
			if (isect.matIdx == -1) {
				bounces--;
				Medium* m = dot(ray.destination, isect.n) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_mediums[isect.mediumOutside])
					: (isect.mediumInside == -1 ? nullptr : &kernel_mediums[isect.mediumInside]);
				ray = Ray(pos, ray.destination, m, kernel_epsilon);

				continue;
			}

			Material mat = kernel_materials[isect.matIdx];

			//direct
			if (!IsDelta(mat.type)) {
				Ray shadowRay;
				float we, cameraPdf;
				int raster;
				kernel_camera->SampleCamera(pos, shadowRay, we, cameraPdf, raster, kernel_epsilon);
				shadowRay.medium = ray.medium;

				if (cameraPdf != 0.f) {
					float3 tr = Tr(shadowRay, uniform, rng);
					float3 fr;
					float unuse;
					Fr(mat, -ray.destination, shadowRay.destination, nor, uv, isect.dpdu, fr, unuse);

					float3 L = tr * beta * fr * we * fabs(dot(shadowRay.destination, nor)) / cameraPdf;
					if (!IsInf(L) && !IsNan(L)) {
						//kernel_color[raster] += L;
						atomicAdd(&kernel_color[raster].x, L.x);
						atomicAdd(&kernel_color[raster].y, L.y);
						atomicAdd(&kernel_color[raster].z, L.z);
					}
				}
			}

			float3 u = make_float3(uniform(rng), uniform(rng), uniform(rng));
			float3 out, fr;
			float pdf;
			SampleBSDF(mat, -ray.destination, nor, uv, isect.dpdu, u, out, fr, pdf, TransportMode::Importance);
			if (IsBlack(fr))
				break;
			beta *= fr * fabs(dot(out, nor)) / pdf;
			Medium* m = dot(out, nor) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_mediums[isect.mediumOutside])
				: (isect.mediumInside == -1 ? nullptr : &kernel_mediums[isect.mediumInside]);
			m = dot(-ray.destination, nor) * dot(out, nor) > 0 ? ray.medium : m;

			ray = Ray(pos, out, m, kernel_epsilon);
		}

		if (bounces > 3) {
			float illumate = clamp(1.f - Luminance(beta), 0.f, 1.f);
			if (uniform(rng) < illumate)
				break;

			beta /= (1 - illumate);
		}
	}
}
//**************************Lighttracing End***********************

//**************************Bdpt Integrator************************
#define BDPT_MAX_DEPTH 65

struct BdptVertex {
	float3 beta;
	Intersection isect;
	Medium* medium = nullptr;
	bool delta;
	float fwd;
	float rev;
};

//convert pdf from area to omega
__device__ float ConvertPdf(float pdf, Intersection& prev, Intersection& cur) {
	float3 dir = prev.pos - cur.pos;
	float square = dot(dir, dir);
	dir = normalize(dir);
	float ret = pdf / square;
	if (!IsBlack(cur.n))
		ret *= fabs(dot(dir, cur.n));
	return ret;
}

__device__ int GenerateCameraPath(int x, int y, BdptVertex* path, thrust::uniform_real_distribution<float>& uniform, thrust::default_random_engine& rng) {
	//start
	float offsetx = uniform(rng) - 0.5f;
	float offsety = uniform(rng) - 0.5f;
	float unuse;
	//bdpt doesn't support dof now
	//float2 aperture = UniformSampleDisk(uniform(rng), uniform(rng), unuse);//for dof
	Ray ray = kernel_camera->GeneratePrimaryRay(x + offsetx, y + offsety, make_float2(0, 0));
	ray.tmin = kernel_epsilon;
	ray.medium = kernel_camera->medium == -1 ? nullptr : &kernel_mediums[kernel_camera->medium];
	float3 beta = { 1.f, 1.f, 1.f };

	int nVertex = 0;
	//set camera isect
	{
		Intersection cameraIsect;
		cameraIsect.pos = kernel_camera->position;
		cameraIsect.n = -kernel_camera->w;
		BdptVertex vertex;
		vertex.beta = beta;
		vertex.isect = cameraIsect;
		vertex.delta = false;
		vertex.medium = ray.medium;
		vertex.fwd = 1.f;
		path[0] = vertex;
	}
	nVertex++;

	float forward = 0.f, rrPdf = 1.f;
	kernel_camera->PdfCamera(ray.destination, unuse, forward);
	Intersection isect;
	int bounces = 0;
	for (; bounces < BDPT_MAX_DEPTH; ++bounces) {
		if (!Intersect(ray, &isect)) {
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.n;
		float2 uv = isect.uv;
		float sampledDist;
		bool sampledMedium = false;
		if (ray.medium) {
			if (ray.medium->type == MT_HOMOGENEOUS)
				beta *= ray.medium->homogeneous.Sample(ray, uniform, rng, sampledDist, sampledMedium);
			else
				beta *= ray.medium->heterogeneous.Sample(ray, uniform, rng, sampledDist, sampledMedium);
		}
		if (IsBlack(beta)) break;
		if (sampledMedium) {
			float3 samplePos = ray(sampledDist);

			float phase, pdf;
			float2 phaseU = make_float2(uniform(rng), uniform(rng));
			float3 dir;
			ray.medium->SamplePhase(phaseU, dir, phase, pdf);
			ray = Ray(samplePos, dir, ray.medium, kernel_epsilon);

			//set medium Intersection
			{
				BdptVertex vertex;
				Intersection mediumIsect;
				mediumIsect.pos = samplePos;
				mediumIsect.n = { 0.f, 0.f, 0.f };
				mediumIsect.matIdx = -1;
				mediumIsect.lightIdx = -1;
				vertex.beta = beta;
				vertex.delta = false;
				vertex.isect = mediumIsect;
				vertex.medium = ray.medium;
				path[bounces + 1] = vertex;
				path[bounces + 1].fwd = ConvertPdf(forward, path[bounces].isect, path[bounces + 1].isect);
				forward = phase;
				path[bounces].rev = ConvertPdf(forward, path[bounces + 1].isect, path[bounces].isect);
			}
			nVertex++;
		}
		else {
			if (isect.matIdx == -1) {
				bounces--;
				Medium* m = dot(ray.destination, isect.n) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_mediums[isect.mediumOutside])
					: (isect.mediumInside == -1 ? nullptr : &kernel_mediums[isect.mediumInside]);
				ray = Ray(pos, ray.destination, m, kernel_epsilon);

				continue;
			}

			Material mat = kernel_materials[isect.matIdx];

			{
				BdptVertex vertex;
				vertex.beta = beta;
				vertex.isect = isect;
				vertex.delta = IsDelta(mat.type);
				vertex.medium = ray.medium;
				path[bounces + 1] = vertex;
				path[bounces + 1].fwd = ConvertPdf(forward, path[bounces].isect, path[bounces + 1].isect);
			}
			nVertex++;

			float3 u = make_float3(uniform(rng), uniform(rng), uniform(rng));
			float3 out, fr;
			float pdf;
			SampleBSDF(mat, -ray.destination, nor, uv, isect.dpdu, u, out, fr, pdf);
			if (IsBlack(fr))
				break;
			beta *= fr * fabs(dot(out, nor)) / pdf;

			forward = pdf;
			if (IsDelta(mat.type)) forward = 0.f;
			//calc reverse pdf
			{
				float3 unuseFr;
				float pdf;
				Fr(mat, out, -ray.destination, nor, uv, isect.dpdu, unuseFr, pdf);
				path[bounces].rev = ConvertPdf(pdf, path[bounces + 1].isect, path[bounces].isect);
			}

			Medium* m = dot(out, nor) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_mediums[isect.mediumOutside])
				: (isect.mediumInside == -1 ? nullptr : &kernel_mediums[isect.mediumInside]);
			m = dot(-ray.destination, nor) * dot(out, nor) > 0 ? ray.medium : m;

			ray = Ray(pos, out, m, kernel_epsilon);
		}

		//russian roulette
		if (bounces > 3) {
			rrPdf = clamp(1.f - Luminance(beta), 0.f, 1.f);
			if (uniform(rng) < rrPdf)
				break;

			beta /= (1 - rrPdf);
		}
	}

	return nVertex;
}

__device__ int GenerateLightPath(BdptVertex* path, thrust::uniform_real_distribution<float>& uniform, thrust::default_random_engine& rng) {
	float3 beta = { 1.f, 1.f, 1.f };
	float choicePdf;
	int lightIdx = LookUpLightDistribution(uniform(rng), choicePdf);
	Area light = kernel_lights[lightIdx];
	float4 u = make_float4(uniform(rng), uniform(rng), uniform(rng), uniform(rng));
	Ray ray;
	float3 lightNor, radiance;
	float pdfA, pdfW;
	light.SampleLight(u, ray, lightNor, radiance, pdfA, pdfW, kernel_epsilon);
	ray.medium = light.medium == -1 ? nullptr : &kernel_mediums[light.medium];

	int nVertex = 0;
	//set light isect
	{
		Intersection lightIsect;
		lightIsect.pos = ray.origin;
		lightIsect.n = lightNor;
		lightIsect.lightIdx = lightIdx;
		BdptVertex vertex;
		vertex.beta = radiance;
		vertex.isect = lightIsect;
		vertex.delta = false;
		vertex.medium = ray.medium;
		vertex.fwd = pdfA * choicePdf;
		path[0] = vertex;
	}
	nVertex++;
	beta *= radiance * fabs(dot(ray.destination, lightNor)) / (pdfA * pdfW * choicePdf);

	Intersection isect;
	float forward = pdfW, rrPdf = 1.f;
	int bounces = 0;
	for (; bounces < BDPT_MAX_DEPTH; ++bounces) {
		if (!Intersect(ray, &isect)) {
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.n;
		float2 uv = isect.uv;
		float sampledDist;
		bool sampledMedium = false;
		if (ray.medium) {
			if (ray.medium->type == MT_HOMOGENEOUS)
				beta *= ray.medium->homogeneous.Sample(ray, uniform, rng, sampledDist, sampledMedium);
			else
				beta *= ray.medium->heterogeneous.Sample(ray, uniform, rng, sampledDist, sampledMedium);
		}
		if (IsBlack(beta)) break;
		if (sampledMedium) {
			float3 samplePos = ray(sampledDist);

			float phase, pdf;
			float2 phaseU = make_float2(uniform(rng), uniform(rng));
			float3 dir;
			ray.medium->SamplePhase(phaseU, dir, phase, pdf);
			ray = Ray(samplePos, dir, ray.medium, kernel_epsilon);

			//set medium Intersection
			{
				BdptVertex vertex;
				Intersection mediumIsect;
				mediumIsect.pos = samplePos;
				mediumIsect.n = { 0.f, 0.f, 0.f };
				mediumIsect.matIdx = -1;
				mediumIsect.lightIdx = -1;
				vertex.beta = beta;
				vertex.delta = false;
				vertex.isect = mediumIsect;
				vertex.medium = ray.medium;
				path[bounces + 1] = vertex;
				path[bounces + 1].fwd = ConvertPdf(forward, path[bounces].isect, path[bounces + 1].isect);
				forward = phase;
				path[bounces].rev = ConvertPdf(phase, path[bounces + 1].isect, path[bounces].isect);
			}
			nVertex++;
		}
		else {
			if (isect.matIdx == -1) {
				bounces--;
				Medium* m = dot(ray.destination, isect.n) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_mediums[isect.mediumOutside])
					: (isect.mediumInside == -1 ? nullptr : &kernel_mediums[isect.mediumInside]);
				ray = Ray(pos, ray.destination, m, kernel_epsilon);

				continue;
			}
			Material mat = kernel_materials[isect.matIdx];

			{
				BdptVertex vertex;
				vertex.beta = beta;
				vertex.isect = isect;
				vertex.delta = IsDelta(mat.type);
				vertex.medium = ray.medium;
				path[bounces + 1] = vertex;
				path[bounces + 1].fwd = ConvertPdf(forward, path[bounces].isect, path[bounces + 1].isect);
			}
			nVertex++;

			float3 u = make_float3(uniform(rng), uniform(rng), uniform(rng));
			float3 out, fr;
			float pdf;
			SampleBSDF(mat, -ray.destination, nor, uv, isect.dpdu, u, out, fr, pdf, TransportMode::Importance);
			if (IsBlack(fr))
				break;
			beta *= fr * fabs(dot(out, nor)) / pdf;

			forward = pdf;
			if (IsDelta(mat.type)) forward = 0.f;
			//calc reverse pdf
			{
				float3 unuseFr;
				float pdf;
				Fr(mat, out, -ray.destination, nor, uv, isect.dpdu, unuseFr, pdf);
				path[bounces].rev = ConvertPdf(pdf, path[bounces + 1].isect, path[bounces].isect);
			}
			Medium* m = dot(out, nor) > 0 ? (isect.mediumOutside == -1 ? nullptr : &kernel_mediums[isect.mediumOutside])
				: (isect.mediumInside == -1 ? nullptr : &kernel_mediums[isect.mediumInside]);
			m = dot(-ray.destination, nor) * dot(out, nor) > 0 ? ray.medium : m;

			ray = Ray(pos, out, m, kernel_epsilon);
		}

		//russian roulette
		if (bounces > 3) {
			rrPdf = clamp(1.f - Luminance(beta), 0.f, 1.f);
			if (uniform(rng) < rrPdf)
				break;

			beta /= (1 - rrPdf);
		}
	}

	return nVertex;
}

__device__ float MisWeight(BdptVertex* cameraPath, BdptVertex* lightPath, int s, int t) {
	if (s + t == 2)//light source is directly visible
		return 1.f;

	//delta bsdf pdf is 0
	auto remap = [](float pdf)->float {
		return pdf == 0 ? 1.f : pdf;
	};

	float sumW = 0.f;
	float ri = 1.f;
	for (int i = s - 1; i > 0; --i) {
		ri *= remap(cameraPath[i].rev) / remap(cameraPath[i].fwd);

		if (!cameraPath[i].delta && !cameraPath[i - 1].delta)
			sumW += ri;
	}

	ri = 1.f;
	for (int i = t - 1; i >= 0; --i) {
		ri *= remap(lightPath[i].rev) / remap(lightPath[i].fwd);

		bool delta = lightPath[i == 0 ? 0 : i - 1].delta;
		if (!lightPath[i].delta && !delta)
			sumW += ri;
	}

	return 1.f / (sumW + 1.f);
}

__device__ float3 Connect(BdptVertex* cameraPath, BdptVertex* lightPath, int s, int t, int& raster,
	thrust::uniform_real_distribution<float>& uniform, thrust::default_random_engine& rng) {
	float3 L = { 0.f, 0.f, 0.f };

	if (t == 0) {
		//naive path tracing
		BdptVertex& cur = cameraPath[s - 1];
		BdptVertex& prev = cameraPath[s - 2];
		if (cur.isect.lightIdx == -1) return{ 0.f, 0.f, 0.f };

		float3 dir = normalize(prev.isect.pos - cur.isect.pos);
		Area light = kernel_lights[cur.isect.lightIdx];
		L += cur.beta * light.Le(cur.isect.n, dir);
		if (IsBlack(L)) return L;

		Ray ray(cur.isect.pos, dir);
		float choicePdf = PdfFromLightDistribution(cur.isect.lightIdx);
		float pdfA, pdfW;
		light.Pdf(ray, cur.isect.n, pdfA, pdfW);
		float curRev = cur.rev;
		float prevRev = prev.rev;
		cur.rev = pdfA * choicePdf;
		prev.rev = ConvertPdf(pdfW, cur.isect, prev.isect);
		float mis = MisWeight(cameraPath, lightPath, s, t);
		//reset
		cur.rev = curRev;
		prev.rev = prevRev;

		return mis * L;
	}
	else if (t == 1) {
		//next event path tracing
		BdptVertex& prev = cameraPath[s - 2];
		BdptVertex& cur = cameraPath[s - 1];
		BdptVertex& next = lightPath[0];
		float3 in = normalize(prev.isect.pos - cur.isect.pos);
		bool isMedium = cur.isect.matIdx == -1;
		Material mat;
		if (!isMedium) mat = kernel_materials[cur.isect.matIdx];
		float choicePdf, lightPdf;
		int idx = LookUpLightDistribution(uniform(rng), choicePdf);
		Area light = kernel_lights[idx];
		float3 radiance, lightNor, lightPos;
		Ray shadowRay;
		float2 lightUniform = { uniform(rng), uniform(rng) };
		light.SampleLight(cur.isect.pos, lightUniform, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);
		lightPos = shadowRay(shadowRay.tmax + kernel_epsilon);
		shadowRay.medium = cur.medium;
		if (IsBlack(radiance)) return{ 0.f, 0.f, 0.f };
		if (!isMedium && IsDelta(mat.type)) return{ 0.f, 0.f, 0.f };
		float3 tr = Tr(shadowRay, uniform, rng);
		if (IsBlack(tr)) return{ 0.f, 0.f, 0.f };

		float3 fr;
		float nextPdf, G, phase;
		if (isMedium) {
			cur.medium->Phase(in, shadowRay.destination, phase, nextPdf);
			fr = make_float3(phase, phase, phase);
			G = 1.f;
		}
		else {
			Fr(mat, in, shadowRay.destination, cur.isect.n, cur.isect.uv, cur.isect.dpdu, fr, nextPdf);
			G = fabs(dot(cur.isect.n, shadowRay.destination));
		}
		L += cur.beta * tr * fr * radiance * G / (lightPdf * choicePdf);
		if (IsBlack(L)) return{ 0.f, 0.f, 0.f };

		BdptVertex tNext = next;
		float pdfA, pdfW;
		light.Pdf(shadowRay, lightNor, pdfA, pdfW);
		next.isect.pos = lightPos;
		next.isect.n = lightNor;
		next.fwd = pdfA * choicePdf;
		next.rev = ConvertPdf(nextPdf, cur.isect, next.isect);
		float curRev = cur.rev;
		float prevRev = prev.rev;
		cur.rev = ConvertPdf(pdfW, next.isect, cur.isect);
		float pdf;
		if (isMedium) pdf = phase;
		else Fr(mat, shadowRay.destination, in, cur.isect.n, cur.isect.uv, cur.isect.dpdu, fr, pdf);
		prev.rev = ConvertPdf(pdf, cur.isect, prev.isect);
		float mis = MisWeight(cameraPath, lightPath, s, t);

		cur.rev = curRev;
		prev.rev = prevRev;
		next = tNext;

		return mis * L;
	}
	else if (s == 1) {
		//light tracing
		BdptVertex& prev = lightPath[t - 2];
		BdptVertex& cur = lightPath[t - 1];
		BdptVertex& next = cameraPath[0];
		float3 in = normalize(prev.isect.pos - cur.isect.pos);
		bool isMedium = cur.isect.matIdx == -1;
		Material mat;
		if (!isMedium) mat = kernel_materials[cur.isect.matIdx];
		Ray shadowRay;
		float we, cameraPdf;
		kernel_camera->SampleCamera(cur.isect.pos, shadowRay, we, cameraPdf, raster, kernel_epsilon);
		shadowRay.medium = cur.medium;
		if (cameraPdf == 0) return{ 0.f, 0.f, 0.f };
		if (!isMedium && IsDelta(mat.type)) return{ 0.f, 0.f, 0.f };
		float3 tr = Tr(shadowRay, uniform, rng);
		if (IsBlack(tr)) return{ 0.f, 0.f, 0.f };

		float3 fr;
		float nextPdf, phase, costheta = fabs(dot(shadowRay.destination, cur.isect.n));
		if (isMedium) {
			cur.medium->Phase(in, shadowRay.destination, phase, nextPdf);
			fr = make_float3(phase, phase, phase);
			costheta = 1.f;
		}
		else Fr(mat, in, shadowRay.destination, cur.isect.n, cur.isect.uv, cur.isect.dpdu, fr, nextPdf);
		L += cur.beta * tr * fr * we * costheta / cameraPdf;
		if (IsBlack(L)) return{ 0.f, 0.f, 0.f };

		float nextRev = next.rev;
		float curRev = cur.rev;
		float prevRev = prev.rev;
		next.rev = ConvertPdf(nextPdf, cur.isect, next.isect);
		float pdfA, pdfW;
		kernel_camera->PdfCamera(-shadowRay.destination, pdfA, pdfW);
		cur.rev = ConvertPdf(pdfW, next.isect, cur.isect);
		float pdf;
		if (isMedium) pdf = phase;
		else Fr(mat, shadowRay.destination, in, cur.isect.n, cur.isect.uv, cur.isect.dpdu, fr, pdf);
		prev.rev = ConvertPdf(pdf, cur.isect, prev.isect);
		float mis = MisWeight(cameraPath, lightPath, s, t);
		next.rev = nextRev;
		cur.rev = curRev;
		prev.rev = prevRev;

		return mis * L;
	}
	else {
		//other
		BdptVertex& c2 = cameraPath[s - 2];
		BdptVertex& c1 = cameraPath[s - 1];
		BdptVertex& l1 = lightPath[t - 1];
		BdptVertex& l2 = lightPath[t - 2];
		float3 l1Tol2 = normalize(l2.isect.pos - l1.isect.pos);
		float3 l1Toc1 = normalize(c1.isect.pos - l1.isect.pos);
		float3 c1Tol1 = -l1Toc1;
		float3 c1Toc2 = normalize(c2.isect.pos - c1.isect.pos);
		float3 dir = c1.isect.pos - l1.isect.pos;
		Material c1Mat, l1Mat;
		if (!c1.medium) c1Mat = kernel_materials[c1.isect.matIdx];
		if (!l1.medium) l1Mat = kernel_materials[l1.isect.matIdx];
		Ray shadowRay;
		shadowRay.origin = c1.isect.pos;
		shadowRay.destination = c1Tol1;
		shadowRay.medium = c1.medium;
		shadowRay.tmin = kernel_epsilon;
		shadowRay.tmax = length(dir) - kernel_epsilon;
		if (!c1.medium && IsDelta(c1Mat.type)) return{ 0.f, 0.f, 0.f };
		if (!l1.medium && IsDelta(l1Mat.type)) return{ 0.f, 0.f, 0.f };
		float3 tr = Tr(shadowRay, uniform, rng);
		if (IsBlack(tr)) return{ 0.f, 0.f, 0.f };
		float cos1 = l1.medium ? 1.f : fabs(dot(l1Toc1, l1.isect.n));
		float cos2 = c1.medium ? 1.f : fabs(dot(c1Tol1, c1.isect.n));

		float3 c1Fr, l1Fr;
		float l1Pdf, c1Pdf;
		float l1Phase, c1Phase;
		if (c1.medium) {
			c1.medium->Phase(c1Toc2, c1Tol1, c1Phase, l1Pdf);
			c1Fr = make_float3(c1Phase, c1Phase, c1Phase);
		}
		else Fr(c1Mat, c1Toc2, c1Tol1, c1.isect.n, c1.isect.uv, c1.isect.dpdu, c1Fr, l1Pdf);
		if (l1.medium) {
			l1.medium->Phase(l1Tol2, l1Toc1, l1Phase, c1Pdf);
			l1Fr = make_float3(l1Phase, l1Phase, l1Phase);
		}
		else Fr(l1Mat, l1Tol2, l1Toc1, l1.isect.n, l1.isect.uv, l1.isect.dpdu, l1Fr, c1Pdf);
		float3 G = tr * cos1 * cos2 / dot(dir, dir);
		L += c1.beta * c1Fr * G * l1Fr * l1.beta;
		if (IsBlack(L)) return{ 0.f, 0.f, 0.f };

		float c2Rev = c2.rev;
		float c1Rev = c1.rev;
		float l1Rev = l1.rev;
		float l2Rev = l2.rev;
		c1.rev = ConvertPdf(c1Pdf, l1.isect, c1.isect);
		l1.rev = ConvertPdf(l1Pdf, c1.isect, l1.isect);
		float l2Pdf, c2Pdf;
		if (l1.medium) l1.medium->Phase(l1Toc1, l1Tol2, l1Phase, l2Pdf);
		else Fr(l1Mat, l1Toc1, l1Tol2, l1.isect.n, l1.isect.uv, l1.isect.dpdu, l1Fr, l2Pdf);
		if (c1.medium) c1.medium->Phase(c1Tol1, c1Toc2, c1Phase, c2Pdf);
		else Fr(c1Mat, c1Tol1, c1Toc2, c1.isect.n, c1.isect.uv, c1.isect.dpdu, c1Fr, c2Pdf);
		l2.rev = ConvertPdf(l2Pdf, l1.isect, l2.isect);
		c2.rev = ConvertPdf(c2Pdf, c1.isect, c2.isect);
		float mis = MisWeight(cameraPath, lightPath, s, t);
		c2.rev = c2Rev;
		c1.rev = c1Rev;
		l1.rev = l1Rev;
		l2.rev = l2Rev;

		return mis * L;
	}

	return L;
}

__global__ void BdptInit() {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	kernel_color[pixel] = { 0.f, 0.f, 0.f };
}

__global__ void Bdpt(int iter, int maxDepth) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	//init seed
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	//too slow to use dynamic allocate
	BdptVertex cameraPath[BDPT_MAX_DEPTH + 2];
	BdptVertex lightPath[BDPT_MAX_DEPTH + 2];
	int nCamera = GenerateCameraPath(x, y, cameraPath, uniform, rng);
	int nLight = GenerateLightPath(lightPath, uniform, rng);
	for (int s = 1; s <= nCamera; ++s) {
		for (int t = 0; t <= nLight; ++t) {
			if ((s == 1 && t == 0) || (s == 1 && t == 1))
				continue;

			int raster;
			float3 L = Connect(cameraPath, lightPath, s, t, raster, uniform, rng);
			if (IsInf(L) || IsNan(L) || IsBlack(L))
				continue;

			if (s == 1) {
				atomicAdd(&kernel_color[raster].x, L.x);
				atomicAdd(&kernel_color[raster].y, L.y);
				atomicAdd(&kernel_color[raster].z, L.z);
				continue;
			}

			atomicAdd(&kernel_color[pixel].x, L.x);
			atomicAdd(&kernel_color[pixel].y, L.y);
			atomicAdd(&kernel_color[pixel].z, L.z);
		}
	}
}
//**************************Bdpt End*******************************

//**************************Mlt Integrator*************************

//gaussian distribution for small step
class MLTSampler {

};

__global__ void Mlt(int iter, int maxDepth) {

}
//**************************Mlt End********************************

//**************************PPM Integrator*************************
struct VisiblePoint {
	float3 ld; //direct light
	float3 ind; //indirect light
	float3 beta; //throughput
	float3 dir;
	Intersection isect;

	float3 tau;
	float radius;
	float n;
	bool valid = false;
};

struct CPUGridNode {
	std::vector<int> vpIdx;
};

VisiblePoint* device_vps;
int* device_vpIdx, * device_vpOffset;
int totalNodes = 0;
__device__ VisiblePoint* vps;
__device__ int* vpIdx, * vpOffset;//grid info
__device__ float3 boundsMin, boundsMax;
__device__ int gridRes[3], hashSize;
__global__ void SPPMSetParam(int* idx, float3 fmin, float3 fmax, int x, int y, int z, int hsize) {
	vpIdx = idx;
	boundsMin = fmin;
	boundsMax = fmax;
	gridRes[0] = x;
	gridRes[1] = y;
	gridRes[2] = z;
	hashSize = hsize;
}

//from pbrt-v3
__host__ __device__ bool ToGrid(float3& p, BBox& bounds, int gridRes[3], float3& pi) {
	bool inBounds = true;
	float3 pg = bounds.Offset(p);
	for (int i = 0; i < 3; ++i) {
		(&pi.x)[i] = (int)(gridRes[i] * (&pg.x)[i]);
		inBounds &= ((&pi.x)[i] >= 0 && (&pi.x)[i] < gridRes[i]);
		(&pi.x)[i] = clamp((int)(&pi.x)[i], 0, gridRes[i] - 1);
	}

	return inBounds;
}

__host__ __device__ unsigned int Hash(int x, int y, int z, int hashSize) {
	//those magic number are some large primes
	return (unsigned int)((x * 73856093) ^ (y * 19349663) ^ (z * 83492791)) % hashSize;
}

//Still too slow, i will be very grateful if someone tells me how to optimize!!
void BuildHashTable(int width, int height) {
	VisiblePoint* host_vps = new VisiblePoint[width * height];
	HANDLE_ERROR(hipMemcpy(host_vps, device_vps, width * height * sizeof(VisiblePoint), hipMemcpyDeviceToHost));

	int hSize = width * height;
	CPUGridNode* grid = new CPUGridNode[hSize];

	BBox gridBounds;
	float initRadius = 0.f;
	for (int i = 0; i < width * height; ++i) {
		gridBounds.Expand(host_vps[i].isect.pos);
		if (host_vps[i].radius > initRadius) initRadius = host_vps[i].radius;
	}

	float3 radius3f = make_float3(initRadius, initRadius, initRadius);
	gridBounds.fmin -= radius3f;
	gridBounds.fmax += radius3f;
	float3 diag = gridBounds.Diagonal();
	float maxDiag = (&diag.x)[gridBounds.GetMaxExtent()];
	int baseGridRes = (int)(maxDiag / initRadius);
	int gRes[3];
	for (int i = 0; i < 3; ++i)
		gRes[i] = Max((int)(baseGridRes * (&diag.x)[i] / maxDiag), 1);

	int total = 0;
	for (int i = 0; i < width * height; ++i) {
		VisiblePoint vp = host_vps[i];
		float3 pMin, pMax;
		ToGrid(vp.isect.pos - radius3f, gridBounds, gRes, pMin);
		ToGrid(vp.isect.pos + radius3f, gridBounds, gRes, pMax);
		for (int z = pMin.z; z <= pMax.z; ++z) {
			for (int y = pMin.y; y <= pMax.y; ++y) {
				for (int x = pMin.x; x <= pMax.x; ++x) {
					int h = Hash(x, y, z, hSize);
					grid[h].vpIdx.push_back(i);
					total++;
				}
			}
		}
	}

	std::vector<int> temp(total), off(hSize + 1); off[0] = 0;
	int* start = &temp[0], offset = 0;
	for (int i = 0; i < hSize; ++i) {
		memcpy(start + offset, &grid[i].vpIdx[0], grid[i].vpIdx.size() * sizeof(int));
		offset += grid[i].vpIdx.size();
		off[i + 1] = offset;
	}

	if (total != totalNodes) {
		HANDLE_ERROR(hipFree(device_vpIdx));
		HANDLE_ERROR(hipMalloc(&device_vpIdx, total * sizeof(int)));
	}
	HANDLE_ERROR(hipMemcpy(device_vpIdx, &temp[0], total * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(device_vpOffset, &off[0], (hSize + 1) * sizeof(int), hipMemcpyHostToDevice));

	SPPMSetParam << <1, 1 >> > (device_vpIdx, gridBounds.fmin, gridBounds.fmax, gRes[0], gRes[1], gRes[2], hSize);
	delete[] host_vps;

	delete[] grid;
}

__device__ void TraceRay(int pixel, Ray r, int iter, int maxDepth, float initRadius, thrust::uniform_real_distribution<float>& uniform, thrust::default_random_engine& rng) {
	VisiblePoint& vp = vps[pixel];
	if (iter == 1) {
		vp.radius = initRadius;
		vp.n = 0.f;
		vp.ld = { 0.f, 0.f, 0.f };
		vp.tau = { 0.f, 0.f, 0.f };
		vp.valid = false;
	}

	float3 beta = { 1.f, 1.f, 1.f };
	Ray ray = r;
	bool specular = false;
	for (int bounces = 0; bounces < maxDepth; ++bounces) {
		Intersection isect;
		if (!Intersect(ray, &isect)) {
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.n;
		float2 uv = isect.uv;
		float3 dpdu = isect.dpdu;
		Material mat = kernel_materials[isect.matIdx];

		float3 Ld = { 0.f, 0.f, 0.f };
		if (!IsDelta(mat.type) && isect.lightIdx == -1) {
			float u = uniform(rng);
			float choicePdf;
			int idx = LookUpLightDistribution(u, choicePdf);
			float2 u1 = make_float2(uniform(rng), uniform(rng));
			float3 radiance, lightNor;
			Ray shadowRay;
			float lightPdf;
			kernel_lights[idx].SampleLight(pos, u1, radiance, shadowRay, lightNor, lightPdf, kernel_epsilon);

			if (!IsBlack(radiance) && !IntersectPrimitive(shadowRay)) {
				float3 fr;
				float samplePdf;

				Fr(mat, -ray.destination, shadowRay.destination, nor, uv, dpdu, fr, samplePdf);

				float weight = PowerHeuristic(1, lightPdf * choicePdf, 1, samplePdf);
				Ld += weight * fr * radiance * fabs(dot(nor, shadowRay.destination)) / (lightPdf * choicePdf);
			}

			float3 us = make_float3(uniform(rng), uniform(rng), uniform(rng));
			float3 out, fr;
			float pdf;
			SampleBSDF(mat, -ray.destination, nor, uv, dpdu, us, out, fr, pdf);
			if (!(IsBlack(fr) || pdf == 0)) {
				Intersection lightIsect;
				Ray lightRay(pos, out, nullptr, kernel_epsilon);
				if (Intersect(lightRay, &lightIsect) && lightIsect.lightIdx != -1) {
					float3 p = lightIsect.pos;
					float3 n = lightIsect.n;
					float3 radiance = { 0.f, 0.f, 0.f };
					radiance = kernel_lights[lightIsect.lightIdx].Le(n, -lightRay.destination);
					if (!IsBlack(radiance)) {
						float pdfA, pdfW;
						kernel_lights[lightIsect.lightIdx].Pdf(Ray(p, -out, nullptr, kernel_epsilon), n, pdfA, pdfW);
						float choicePdf = PdfFromLightDistribution(lightIsect.lightIdx);
						float lenSquare = dot(p - pos, p - pos);
						float costheta = fabs(dot(n, lightRay.destination));
						float lPdf = pdfA * lenSquare / (costheta);
						float weight = PowerHeuristic(1, pdf, 1, lPdf * choicePdf);

						Ld += weight * fr * radiance * fabs(dot(out, nor)) / pdf;
					}
				}

			}
		}

		//light vp
		if (bounces == 0 || (specular && isect.lightIdx != -1)) {
			Ld += kernel_lights[isect.lightIdx].Le(nor, -ray.destination);
		}

		if (!IsNan(Ld)) vp.ld += beta * Ld;

		//delta material should be more careful
		if (IsDelta(mat.type) || (IsGlossy(mat.type) && mat.alphaU < 0.2f)) {
			float3 fr, out;
			float pdf;
			float3 uniformBsdf = make_float3(uniform(rng), uniform(rng), uniform(rng));
			SampleBSDF(mat, -ray.destination, nor, uv, dpdu, uniformBsdf, out, fr, pdf);
			if (IsBlack(fr)) return;

			beta *= fr * fabs(dot(out, nor)) / pdf;
			specular = IsDelta(mat.type);

			ray = Ray(pos, out, nullptr, kernel_epsilon);

			continue;
		}

		vp.beta = beta;
		vp.dir = -ray.destination;
		vp.isect = isect;
		vp.valid = true;

		break;
	};
}

__device__ void TracePhoton(int maxDepth, thrust::uniform_real_distribution<float>& uniform, thrust::default_random_engine& rng) {
	float3 beta = { 1.f, 1.f, 1.f };
	float choicePdf;
	int idx = LookUpLightDistribution(uniform(rng), choicePdf);
	Area light = kernel_lights[idx];
	float3 radiance, lightNor;
	float4 lightUniform = { uniform(rng), uniform(rng), uniform(rng), uniform(rng) };
	Ray ray;
	float pdfA, pdfW;
	light.SampleLight(lightUniform, ray, lightNor, radiance, pdfA, pdfW, kernel_epsilon);
	beta *= radiance * fabs(dot(lightNor, ray.destination)) / (pdfA * pdfW * choicePdf);

	Intersection isect;
	for (int bounces = 0; bounces < maxDepth; ++bounces) {
		if (!Intersect(ray, &isect)) {
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.n;
		float2 uv = isect.uv;
		float3 dpdu = isect.dpdu;
		Material mat = kernel_materials[isect.matIdx];
		if (bounces > 0) {//bounces = 0 are already taken into account
			float3 gridCoord;
			BBox gridBounds(boundsMin, boundsMax);
			if (ToGrid(pos, gridBounds, gridRes, gridCoord)) {
				int h = Hash(gridCoord.x, gridCoord.y, gridCoord.z, hashSize);
				int start = vpOffset[h];
				int vpSize = vpOffset[h + 1] - start;
				for (int i = 0; i < vpSize; ++i) {
					int idx = vpIdx[start + i];
					VisiblePoint& vp = vps[idx];
					if (!vp.valid) continue;
					float3 out = pos - vp.isect.pos;
					float distanceSquare = dot(out, out);
					if (distanceSquare > vp.radius * vp.radius) continue;
					Material vpMat = kernel_materials[vp.isect.matIdx];
					float3 fr;
					float pdf;
					Fr(vpMat, vp.dir, -ray.destination, vp.isect.n, vp.isect.uv, vp.isect.dpdu, fr, pdf);
					if (IsBlack(fr) || IsNan(fr)) continue;
					float3 b = fr * beta * vp.beta;
					b += vp.tau;

					//suppose just a photon hit the same visible point at the same time
					float alpha = 0.7f;
					float g = (vp.n + alpha) / (vp.n + 1.f);
					float rnew = vp.radius * sqrt(g);
					vp.tau = b * g;
					vp.n += alpha;
					vp.radius = rnew;
				}
			}
		}

		float3 fr, out;
		float3 bsdfUniform = make_float3(uniform(rng), uniform(rng), uniform(rng));
		float pdf;
		SampleBSDF(mat, -ray.destination, nor, uv, dpdu, bsdfUniform, out, fr, pdf, TransportMode::Importance);
		if (pdf == 0) break;

		beta *= fr * fabs(dot(nor, out)) / pdf;

		ray = Ray(pos, out, nullptr, kernel_epsilon);

		if (bounces > 3) {
			float illumate = clamp(1.f - Luminance(beta), 0.f, 1.f);
			if (uniform(rng) < illumate)
				break;

			beta /= (1 - illumate);
		}
	}
}

__global__ void StochasticProgressivePhotonmapperInit(VisiblePoint* v, int* offset) {
	vps = v;
	vpOffset = offset;
}

//first pass trace eye ray
__global__ void StochasticProgressivePhotonmapperFP(int iter, int maxDepth, float initRadius = 0.5f) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	//init seed
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	float offsetx = uniform(rng) - 0.5f;
	float offsety = uniform(rng) - 0.5f;
	float unuse;
	//ppm doesn't support dof now
	//float2 aperture = UniformSampleDisk(uniform(rng), uniform(rng), unuse);//for dof
	Ray ray = kernel_camera->GeneratePrimaryRay(x + offsetx, y + offsety, make_float2(0, 0));
	ray.tmin = kernel_epsilon;

	TraceRay(pixel, ray, iter, maxDepth, initRadius, uniform, rng);
}

//build hash table for vp
void StochasticProgressivePhotonmapperBuildHashTable(int width, int height) {
	BuildHashTable(width, height);
}

//second pass trace photon
__global__ void StochasticProgressivePhotonmapperSP(int iter, int maxDepth) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	//init seed
	int threadIndex = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter * iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	TracePhoton(maxDepth, uniform, rng);
}

//third pass density evaluate
__global__ void StochasticProgressivePhotonmapperTP(int iter, int photonsPerIteration) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	VisiblePoint& vp = vps[pixel];

	float3 L = { 0.f, 0.f, 0.f };
	if (vp.valid) {
		//as the number of iterations increases, the radius becomes
		//smaller and samller, eventually producing infinity indirect
		float3 indirect = vp.tau / (PI * vp.radius * vp.radius * photonsPerIteration * iter);
		//skip if color is not a number
		if (IsNan(indirect) || IsInf(indirect)) indirect = vp.ind;
		vp.ind = indirect;
		L = vp.ld / iter + indirect;
	}
	kernel_color[pixel] = L;
}
//**************************SPPM End********************************

//**************************Instant Radiosity Integrator************
#define IR_MAX_VPLS 32
struct Vpl {
	float3 beta;
	float3 dir;
	float3 pos;
	float3 nor;
	float2 uv;
	float3 dpdu;
	int matIdx;
};

__device__ Vpl vpls[IR_MAX_VPLS][IR_MAX_VPLS];
__device__ int numVpls[IR_MAX_VPLS];
int vplIter = IR_MAX_VPLS;

__global__ void GenerateVpl(int iter, int maxDepth) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	numVpls[pixel] = 0;
	float3 beta = { 1.f, 1.f, 1.f };
	float choicePdf;
	int idx = LookUpLightDistribution(uniform(rng), choicePdf);
	Area light = kernel_lights[idx];
	float3 radiance, lightNor;
	Ray ray;
	float4 lightUniform = make_float4(uniform(rng), uniform(rng), uniform(rng), uniform(rng));
	float pdfA, pdfW;
	light.SampleLight(lightUniform, ray, lightNor, radiance, pdfA, pdfW, kernel_epsilon);
	beta *= radiance * fabs(dot(lightNor, ray.destination)) / (pdfA * pdfW * choicePdf);
	{
		Vpl vpl;
		vpl.beta = radiance;
		vpl.dir.x = pdfA * choicePdf;
		vpl.pos = ray.origin;
		vpl.nor = lightNor;
		vpls[pixel][numVpls[pixel]++] = vpl;
	}

	Intersection isect;
	for (int bounces = 0; bounces < maxDepth; ++bounces) {
		if (!Intersect(ray, &isect)) {
			break;
		}

		float3 pos = isect.pos;
		float3 nor = isect.n;
		float2 uv = isect.uv;
		float3 dpdu = isect.dpdu;
		Material mat = kernel_materials[isect.matIdx];

		{
			Vpl vpl;
			vpl.beta = beta;
			vpl.dir = -ray.destination;
			vpl.pos = isect.pos;
			vpl.nor = isect.n;
			vpl.uv = isect.uv;
			vpl.dpdu = isect.dpdu;
			vpl.matIdx = isect.matIdx;
			vpls[pixel][numVpls[pixel]++] = vpl;
		}

		float3 fr, out;
		float3 bsdfUniform = make_float3(uniform(rng), uniform(rng), uniform(rng));
		float pdf;
		SampleBSDF(mat, -ray.destination, nor, uv, dpdu, bsdfUniform, out, fr, pdf, TransportMode::Importance);
		if (IsBlack(fr)) break;

		beta *= fr * fabs(dot(out, nor)) / pdf;

		ray = Ray(pos, out, nullptr, kernel_epsilon);

		if (bounces > 3) {
			float illumate = clamp(1.f - Luminance(beta), 0.f, 1.f);
			if (uniform(rng) < illumate)
				break;

			beta /= (1 - illumate);
		}
	}
}

__global__ void InstantRadiosity(int iter, int vplIter, int maxDepth, float bias) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	thrust::default_random_engine rng(WangHash(pixel) + WangHash(iter));
	thrust::uniform_real_distribution<float> uniform(0.0f, 1.0f);

	float offsetx = uniform(rng) - 0.5f;
	float offsety = uniform(rng) - 0.5f;
	float unuse;
	float2 aperture = UniformSampleDisk(uniform(rng), uniform(rng), unuse);//for dof
	Ray ray = kernel_camera->GeneratePrimaryRay(x + offsetx, y + offsety, aperture);
	ray.tmin = kernel_epsilon;
	float3 beta = { 1.f, 1.f, 1.f };
	float3 L = { 0.f, 0.f, 0.f };

	Intersection isect;
	for (int bounces = 0; bounces < maxDepth; ++bounces) {
		if (!Intersect(ray, &isect)) break;
		if (isect.lightIdx != -1) {
			L += kernel_lights[isect.lightIdx].Le(isect.n, -ray.destination);
		}
		float3 pos = isect.pos;
		float3 nor = isect.n;
		float2 uv = isect.uv;
		float3 dpdu = isect.dpdu;
		Material mat = kernel_materials[isect.matIdx];
		if (IsDelta(mat.type)) {
			float3 fr, out;
			float3 bsdfUniform = make_float3(uniform(rng), uniform(rng), uniform(rng));
			float pdf;
			SampleBSDF(mat, -ray.destination, nor, uv, dpdu, bsdfUniform, out, fr, pdf);
			if (IsBlack(fr)) break;
			beta *= fr * fabs(dot(nor, out)) / pdf;

			ray = Ray(pos, out, nullptr, kernel_epsilon);
			continue;
		}

		for (int i = 0; i < numVpls[vplIter]; ++i) {
			Vpl vpl = vpls[vplIter][i];

			float3 dir = pos - vpl.pos;
			float3 out = normalize(dir);
			float squreDistance = dot(dir, dir);
			Ray shadowRay(pos, -out, nullptr, kernel_epsilon, sqrt(squreDistance) - kernel_epsilon);
			if (IntersectPrimitive(shadowRay)) continue;

			if (squreDistance < bias) squreDistance = bias;
			float c1 = fabs(dot(out, nor));
			float c2 = fabs(dot(out, vpl.nor));
			float G = c1 * c2 / squreDistance;
			float3 fr1, fr2;
			float pdf1, pdf2;
			Fr(mat, -ray.destination, -out, nor, uv, dpdu, fr1, pdf1);
			if (i == 0) {
				if (dot(dir, vpl.nor) > 0.f)
					L += beta * fr1 * G * vpl.beta / vpl.dir.x;
				continue;
			}
			Material m = kernel_materials[vpl.matIdx];
			if (IsDelta(m.type)) continue;
			Fr(m, vpl.dir, out, vpl.nor, vpl.uv, vpl.dpdu, fr2, pdf2);

			L += beta * fr1 * G * fr2 * vpl.beta;
		}

		break;
	}

	if (IsNan(L) || IsInf(L)) return;

	kernel_color[pixel] = L;
}
//**************************Instant Radiosity Integrator End********

__global__ void Output(int iter, float3* output, bool reset, bool filmic, IntegratorType type) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned pixel = x + y * blockDim.x * gridDim.x;

	if (reset) kernel_acc_image[pixel] = { 0, 0, 0 };

	float3 color = kernel_color[pixel];
	if (type != IT_SPPM) {
		kernel_acc_image[pixel] += color;
		color = kernel_acc_image[pixel] / iter;
	}
	if (filmic) FilmicTonemapping(color);
	else GammaCorrection(color);
	output[pixel] = color;
}

__global__ void InitRender(
	Camera* camera,
	LinearBVHNode* bvh_nodes,
	Primitive* primitives,
	Material* materials,
	Bssrdf* bssrdfs,
	Medium* mediums,
	Area* lights,
	Infinite* infinite,
	uchar4** texs,
	float* light_distribution,
	int light_size,
	int ld_size,
	int* tex_size,
	float3* image,
	float3* color,
	float ep) {
	kernel_camera = camera;
	kernel_linear = bvh_nodes;
	kernel_primitives = primitives;
	kernel_materials = materials;
	kernel_bssrdfs = bssrdfs;
	kernel_mediums = mediums;
	kernel_lights = lights;
	kernel_infinite = infinite;
	kernel_textures = texs;
	kernel_light_distribution = light_distribution;
	kernel_light_size = light_size;
	kernel_light_distribution_size = ld_size;
	kernel_texture_size = tex_size;
	kernel_acc_image = image;
	kernel_color = color;
	kernel_epsilon = ep;
}

void BeginRender(
	Scene& scene,
	unsigned width,
	unsigned height,
	float ep) {
	int mesh_memory_use = 0;
	int material_memory_use = 0;
	int bvh_memory_use = 0;
	int light_memory_use = 0;
	int texture_memory_use = 0;
	int num_primitives = scene.bvh.prims.size();
	HANDLE_ERROR(hipMalloc(&dev_camera, sizeof(Camera)));
	HANDLE_ERROR(hipMemcpy(dev_camera, scene.camera, sizeof(Camera), hipMemcpyHostToDevice));

	if (num_primitives) {
		HANDLE_ERROR(hipMalloc(&dev_primitives, num_primitives * sizeof(Primitive)));
		HANDLE_ERROR(hipMemcpy(dev_primitives, &scene.bvh.prims[0], num_primitives * sizeof(Primitive), hipMemcpyHostToDevice));
		mesh_memory_use += num_primitives * sizeof(Primitive);
	}
	if (scene.bvh.total_nodes > 0) {
		HANDLE_ERROR(hipMalloc(&dev_bvh_nodes, scene.bvh.total_nodes * sizeof(LinearBVHNode)));
		HANDLE_ERROR(hipMemcpy(dev_bvh_nodes, scene.bvh.linear_root, scene.bvh.total_nodes * sizeof(LinearBVHNode), hipMemcpyHostToDevice));
		bvh_memory_use += scene.bvh.total_nodes * sizeof(LinearBVHNode);
	}

	//copy material
	int num_materials = scene.materials.size();
	HANDLE_ERROR(hipMalloc(&dev_materials, num_materials * sizeof(Material)));
	HANDLE_ERROR(hipMemcpy(dev_materials, &scene.materials[0], num_materials * sizeof(Material), hipMemcpyHostToDevice));
	material_memory_use += num_materials * sizeof(Material);

	int num_bssrdfs = scene.bssrdfs.size();
	if (num_bssrdfs) {
		HANDLE_ERROR(hipMalloc(&dev_bssrdfs, num_bssrdfs * sizeof(Bssrdf)));
		HANDLE_ERROR(hipMemcpy(dev_bssrdfs, &scene.bssrdfs[0], num_bssrdfs * sizeof(Bssrdf), hipMemcpyHostToDevice));
		material_memory_use += num_bssrdfs * sizeof(Bssrdf);
	}

	int num_mediums = scene.mediums.size();
	if (num_mediums) {
		HANDLE_ERROR(hipMalloc(&dev_mediums, num_mediums * sizeof(Medium)));
		HANDLE_ERROR(hipMemcpy(dev_mediums, &scene.mediums[0], num_mediums * sizeof(Medium), hipMemcpyHostToDevice));
		material_memory_use += num_mediums * sizeof(Medium);
	}
	//copy heterogeneous density data
	for (int i = 0; i < num_mediums; ++i) {
		if (scene.mediums[i].type == MT_HETEROGENEOUS) {
			Heterogeneous m = scene.mediums[i].heterogeneous;
			float* density;
			HANDLE_ERROR(hipMalloc(&density, m.nx * m.ny * m.nz * sizeof(float)));
			material_memory_use += m.nx * m.ny * m.nz * sizeof(float);
			HANDLE_ERROR(hipMemcpy(density, m.density, m.nx * m.ny * m.nz * sizeof(float), hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(&dev_mediums[i].heterogeneous.density, &density, sizeof(float*), hipMemcpyHostToDevice));
			delete[] m.density;
		}
	}

	//copy light
	int num_lights = scene.lights.size();
	if (num_lights) {
		HANDLE_ERROR(hipMalloc(&dev_lights, num_lights * sizeof(Area)));
		HANDLE_ERROR(hipMemcpy(dev_lights, &scene.lights[0], num_lights * sizeof(Area), hipMemcpyHostToDevice));
		light_memory_use += num_lights * sizeof(Area);
	}

	//copy infinite light
	HANDLE_ERROR(hipMalloc(&dev_infinite, sizeof(Infinite)));
	HANDLE_ERROR(hipMemcpy(dev_infinite, &scene.infinite, sizeof(Infinite), hipMemcpyHostToDevice));
	if (scene.infinite.isvalid) {
		int width = scene.infinite.width, height = scene.infinite.height;
		float3* data;
		HANDLE_ERROR(hipMalloc(&data, width * height * sizeof(float3)));
		texture_memory_use += width * height * sizeof(float3);
		HANDLE_ERROR(hipMemcpy(data, scene.infinite.data, width * height * sizeof(float3), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(&dev_infinite->data, &data, sizeof(float3*), hipMemcpyHostToDevice));
	}

	//copy textures
	if (scene.textures.size()) {
		HANDLE_ERROR(hipMalloc(&texture_size, scene.textures.size() * 2 * sizeof(int)));
		std::vector<int> texSize;
		HANDLE_ERROR(hipMalloc(&dev_textures, scene.textures.size() * sizeof(uchar4*)));
		for (int i = 0; i < scene.textures.size(); ++i) {
			Texture tex = scene.textures[i];
			texSize.push_back(tex.width);
			texSize.push_back(tex.height);
			uchar4* t;
			HANDLE_ERROR(hipMalloc(&t, tex.width * tex.height * sizeof(uchar4)));
			texture_memory_use += tex.width * tex.height * sizeof(uchar4);
			HANDLE_ERROR(hipMemcpy(t, &tex.data[0], tex.width * tex.height * sizeof(uchar4), hipMemcpyHostToDevice));
			HANDLE_ERROR(hipMemcpy(&dev_textures[i], &t, sizeof(uchar4*), hipMemcpyHostToDevice));
		}
		HANDLE_ERROR(hipMemcpy(texture_size, &texSize[0], scene.textures.size() * 2 * sizeof(int), hipMemcpyHostToDevice));
	}

	int num_pixel = width * height;
	HANDLE_ERROR(hipMalloc(&dev_image, num_pixel * sizeof(float3)));
	texture_memory_use += num_pixel * sizeof(float3);
	HANDLE_ERROR(hipMalloc(&dev_color, num_pixel * sizeof(float3)));
	texture_memory_use += num_pixel * sizeof(float3);

	int ld_size = scene.lightDistribution.size();
	HANDLE_ERROR(hipMalloc(&dev_light_distribution, ld_size * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(dev_light_distribution, &scene.lightDistribution[0], ld_size * sizeof(float), hipMemcpyHostToDevice));
	texture_memory_use += ld_size * sizeof(float);

	InitRender << <1, 1 >> > (dev_camera, dev_bvh_nodes,
		dev_primitives, dev_materials, dev_bssrdfs, dev_mediums, dev_lights, dev_infinite, dev_textures, dev_light_distribution, num_lights, ld_size,
		texture_size, dev_image, dev_color, ep);

	//init for progressive photon mapper
	if (scene.integrator.type == IT_SPPM) {
		HANDLE_ERROR(hipMalloc(&device_vps, width * height * sizeof(VisiblePoint)));
		HANDLE_ERROR(hipMalloc(&device_vpOffset, (width * height + 1) * sizeof(int)));
		HANDLE_ERROR(hipMalloc(&device_vpIdx, sizeof(int)));

		StochasticProgressivePhotonmapperInit << <1, 1 >> > (device_vps, device_vpOffset);
	}

	HANDLE_ERROR(hipDeviceSynchronize());

	fprintf(stderr, "\n\nMesh video memory use:[%.3fM]\n", (float)mesh_memory_use / (1024 * 1024));
	fprintf(stderr, "Bvh video memory use:[%.3fM]\n", (float)bvh_memory_use / (1024 * 1024));
	fprintf(stderr, "Material video memory use:[%.3fM]\n", (float)material_memory_use / (1024 * 1024));
	fprintf(stderr, "Light video memory use:[%.3fM]\n", (float)light_memory_use / (1024 * 1024));
	fprintf(stderr, "Texture video memory use:[%.2fM]\n", (float)texture_memory_use / (1024 * 1024));
	fprintf(stderr, "Total video memory use:[%.3fM]\n", (float)(mesh_memory_use + bvh_memory_use + material_memory_use + light_memory_use + texture_memory_use) / (1024 * 1024));
}

void EndRender() {
	HANDLE_ERROR(hipFree(dev_primitives));
	HANDLE_ERROR(hipFree(dev_bvh_nodes));

	HANDLE_ERROR(hipFree(dev_image));
	HANDLE_ERROR(hipFree(dev_color));
}

void Render(Scene& scene, unsigned width, unsigned height, Camera* camera, unsigned iter, bool reset, float3* output) {
	//HANDLE_ERROR(hipMemcpy(dev_camera, camera, sizeof(Camera), hipMemcpyHostToDevice));
	int block_x = 32, block_y = 4;
	dim3 block(block_x, block_y);
	dim3 grid(width / block.x, height / block.y);

	IntegratorType type = scene.integrator.type;
	if (type == IT_AO)
		Ao << <grid, block >> > (iter, scene.integrator.maxDist);
	else if (type == IT_PT)
		Path << <grid, block >> > (iter, scene.integrator.maxDepth);
	else if (type == IT_VPT)
		Volpath << <grid, block >> > (iter, scene.integrator.maxDepth);
	else if (type == IT_LT) {
		LightTracingInit << <grid, block >> > ();
		LightTracing << <grid, block >> > (iter, scene.integrator.maxDepth);
	}
	else if (type == IT_BDPT) {
		BdptInit << <grid, block >> > ();
		Bdpt << <grid, block >> > (iter, scene.integrator.maxDepth);
	}
	else if (type == IT_SPPM) {
		StochasticProgressivePhotonmapperFP << <grid, block >> > (iter, scene.integrator.maxDepth,
			scene.integrator.initRadius);

		//build hash grid on cpu
		StochasticProgressivePhotonmapperBuildHashTable(width, height);

		int photonsPerIteration = scene.integrator.photonsPerIteration;
		StochasticProgressivePhotonmapperSP << < photonsPerIteration / 10, 10 >> > (iter, scene.integrator.maxDepth);

		StochasticProgressivePhotonmapperTP << <grid, block >> > (iter, photonsPerIteration);
	}
	else if (type == IT_IR) {
		if (vplIter == IR_MAX_VPLS) {
			vplIter = 0;
			GenerateVpl << <IR_MAX_VPLS, 1 >> > (iter, scene.integrator.maxDepth);
		}
		InstantRadiosity << <grid, block >> > (iter, vplIter, scene.integrator.maxDepth, scene.integrator.vplBias);
		vplIter++;
	}

	grid.x = width / block.x;
	grid.y = height / block.y;
	Output << <grid, block >> > (iter, output, reset, camera->filmic, type);
}